/**********************************************************************************/
/* MIT License                                                                    */
/*                                                                                */
/* Copyright (c) 2020, 2021 JetBrains-Research                                    */
/*                                                                                */
/* Permission is hereby granted, free of charge, to any person obtaining a copy   */
/* of this software and associated documentation files (the "Software"), to deal  */
/* in the Software without restriction, including without limitation the rights   */
/* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      */
/* copies of the Software, and to permit persons to whom the Software is          */
/* furnished to do so, subject to the following conditions:                       */
/*                                                                                */
/* The above copyright notice and this permission notice shall be included in all */
/* copies or substantial portions of the Software.                                */
/*                                                                                */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     */
/* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       */
/* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    */
/* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         */
/* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  */
/* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  */
/* SOFTWARE.                                                                      */
/**********************************************************************************/

#include <cuda/cuda_matrix.hpp>
#include <cuda/kernels/sptranspose.cuh>
#include <cuda/kernels/sptranspose2.cuh>

namespace spbla {

    void CudaMatrix::transpose(const MatrixBase &otherBase, bool checkTime) {
        auto other = dynamic_cast<const CudaMatrix*>(&otherBase);

        CHECK_RAISE_ERROR(other != nullptr, InvalidArgument, "Passed matrix does not belong to csr matrix class");

        size_t M = other->getNrows();
        size_t N = other->getNcols();

        assert(this->getNrows() == N);
        assert(this->getNcols() == M);

        kernels::SpTranspose2Functor<index, DeviceAlloc<index>> spTranspose2Functor;
        auto result = spTranspose2Functor(other->mMatrixImpl);

        // Assign the actual impl result to this storage
        this->mMatrixImpl = std::move(result);
    }

}