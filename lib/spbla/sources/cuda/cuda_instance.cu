/**********************************************************************************/
/* MIT License                                                                    */
/*                                                                                */
/* Copyright (c) 2020, 2021 JetBrains-Research                                    */
/*                                                                                */
/* Permission is hereby granted, free of charge, to any person obtaining a copy   */
/* of this software and associated documentation files (the "Software"), to deal  */
/* in the Software without restriction, including without limitation the rights   */
/* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      */
/* copies of the Software, and to permit persons to whom the Software is          */
/* furnished to do so, subject to the following conditions:                       */
/*                                                                                */
/* The above copyright notice and this permission notice shall be included in all */
/* copies or substantial portions of the Software.                                */
/*                                                                                */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     */
/* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       */
/* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    */
/* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         */
/* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  */
/* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  */
/* SOFTWARE.                                                                      */
/**********************************************************************************/

#include <cuda/cuda_instance.hpp>
#include <core/error.hpp>
#include <string>
#include <cstring>
#include <cassert>
#include <cstdio>

namespace spbla {

    CudaInstance::~CudaInstance() {
        assert(mHostAllocCount == 0);
        assert(mDeviceAllocCount == 0);

        gInstance = nullptr;
    }

    void CudaInstance::allocateOnGpu(void* &ptr, size_t size) const {
        hipError_t error;

        switch (mMemoryType) {
            case MemType::Default:
                error = hipMalloc(&ptr, size);
                break;
            case MemType::Managed:
                error = hipMallocManaged(&ptr, size);
                break;
            default:
                RAISE_ERROR(MemOpFailed, "Failed to fined suitable allocator");
        }

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to allocate Gpu memory: "} + hipGetErrorString(error);
            RAISE_ERROR(MemOpFailed, message);
        }

        mDeviceAllocCount++;
    }

    void CudaInstance::deallocateOnGpu(void* ptr) const {
        hipError_t error = hipFree(ptr);

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to deallocate Gpu memory: "} + hipGetErrorString(error);
            RAISE_ERROR(MemOpFailed, message);
        }

        mDeviceAllocCount--;
    }

    void CudaInstance::syncHostDevice() const {
        hipError_t error = hipDeviceSynchronize();

        if (error != hipSuccess) {
            std::string message = std::string{"Failed to synchronize host and device: "} + hipGetErrorString(error);
            RAISE_ERROR(DeviceError, message);
        }
    }

    bool CudaInstance::isCudaDeviceSupported() {
        int device;
        hipError_t error = hipGetDevice(&device);
        return error == hipSuccess;
    }

    void CudaInstance::queryDeviceCapabilities(spbla_DeviceCaps &deviceCaps) {
        const unsigned long long KiB = 1024;

        int device;
        hipError_t error = hipGetDevice(&device);

        if (error == hipSuccess) {
            hipDeviceProp_t deviceProp{};
            error = hipGetDeviceProperties(&deviceProp, device);

            if (error == hipSuccess) {
                std::snprintf(deviceCaps.name, sizeof(deviceCaps.name), "%s", deviceProp.name);
                deviceCaps.cudaSupported = true;
                deviceCaps.minor = deviceProp.minor;
                deviceCaps.major = deviceProp.major;
                deviceCaps.warp = deviceProp.warpSize;
                deviceCaps.globalMemoryKiBs = deviceProp.totalGlobalMem / KiB;
                deviceCaps.sharedMemoryPerMultiProcKiBs = deviceProp.sharedMemPerMultiprocessor / KiB;
                deviceCaps.sharedMemoryPerBlockKiBs = deviceProp.sharedMemPerBlock / KiB;
            }
        }
    }

}
