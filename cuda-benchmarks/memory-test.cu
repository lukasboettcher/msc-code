
__global__ void kernel(size_t *A, size_t N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        A[i] = i;
}

#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char *const func, const char *const file, const int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error at: '%s:%i'\n\t%s %s\n", file, line, hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}

void verify(size_t N, size_t *d_A)
{
    for (size_t i = 0; i < N; i++)
    {
        // assert(h_A[i] == i);
        if (d_A[i] != i)
        {
            printf("%lu != %lu\n", d_A[i], i);
        }
    }
}

void test_managed()
{
    hipDeviceProp_t prop; // CUDA device properties variable
    checkCuda(hipGetDeviceProperties(&prop, 0));
    // size_t N = prop.totalGlobalMem - 1024 * 1024 * 1024;
    size_t N = 10 * 1024 * 1024 * 1024L;
    size_t N_entries = N / sizeof(size_t);
    size_t *d_A;

    checkCuda(hipMallocManaged(&d_A, N));
    memset(d_A, UCHAR_MAX, N);

    // size_t *h_A = (size_t *)malloc(N);
    // memset(h_A, UCHAR_MAX, N);

    // checkCuda(cudaMalloc(&d_A, N));
    // checkCuda(cudaMemcpy(d_A, h_A, N, cudaMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, N_entries);
    checkCuda(hipDeviceSynchronize());

    // checkCuda(cudaMemcpy(h_A, d_A, N, cudaMemcpyDeviceToHost));
    verify(N_entries, d_A);

    checkCuda(hipFree(d_A));
    // free(h_A);
};

void test_native()
{
    hipDeviceProp_t prop; // CUDA device properties variable
    checkCuda(hipGetDeviceProperties(&prop, 0));
    // size_t N = prop.totalGlobalMem - 1024 * 1024 * 1024;
    size_t N = 10 * 1024 * 1024 * 1024L;
    size_t N_entries = N / sizeof(size_t);
    size_t *d_A;

    size_t *h_A = (size_t *)malloc(N);
    memset(h_A, UCHAR_MAX, N);

    checkCuda(hipMalloc(&d_A, N));
    checkCuda(hipMemcpy(d_A, h_A, N, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, N_entries);
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipMemcpy(h_A, d_A, N, hipMemcpyDeviceToHost));
    verify(N_entries, h_A);

    checkCuda(hipFree(d_A));
    free(h_A);
};

int main(int argc, char const *argv[])
{
    test_managed();
    // test_native();
    return 0;
}
