#include "hip/hip_runtime.h"
#include <pthread.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <time.h>
#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char *const func, const char *const file, const int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error at: '%s:%i'\n\t%s %s\n", file, line, hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}

int args[100];
// const size_t N = 1 * 1024 * 1024 * 1024L;
const size_t N = 2 * 1024 * 1024 * 1024L;
int num_gpus;

__device__ __managed__ uint *__memory__;
__managed__ uint N_FIB = 10000;

__device__ __host__ uint fib(uint n)
{
    uint a = 0, b = 1, c, i;
    if (n == 0)
        return a;
    for (i = 2; i <= n; i++)
    {
        c = a + b;
        a = b;
        b = c;
    }
    return b;
}

__global__ void kernel(size_t start, size_t end)
{
    uint tid = start + threadIdx.x + blockIdx.x * blockDim.x;
    for (uint i = tid; i < end; i += blockDim.x * gridDim.x)
    {
        __memory__[i] = i + fib(N_FIB);
    }
}

void *launch_kernel(void *arg)
{
    int threadId = *((int *)arg);

    hipSetDevice(threadId);

    size_t perGpu = (N + num_gpus - 1) / num_gpus;

    size_t start = threadId * perGpu;
    size_t end = min(N, start + perGpu);

    hipMemAdvise(__memory__ + start, perGpu, hipMemAdviseSetPreferredLocation, threadId);
    hipMemPrefetchAsync(__memory__ + start, perGpu, threadId, hipStreamPerThread);

    printf("\tstarting thread %i w/ start: %lu end: %lu | total: %lu, pergpu: %lu\n", threadId, start, end, N, perGpu);

    kernel<<<80, 1024>>>(start, end);

    hipStreamSynchronize(hipStreamPerThread);

    return NULL;
}

void verify()
{
    uint static_fib = fib(N_FIB);
    for (size_t i = 0; i < N; i++)
        // assert(__memory__[i] == i);
        if (__memory__[i] != i + static_fib)
        {
            fprintf(stderr, "error, w/ iter: %lu\n", i);
            break;
        }
}

void run_multi_kernel()
{
    hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int num_threads = num_gpus;

    pthread_t threads[num_threads];

    hipEventRecord(start, 0);

    for (int i = 0; i < num_threads; i++)
    {
        args[i] = i;
        if (pthread_create(&threads[i], NULL, launch_kernel, &args[i]))
        {
            fprintf(stderr, "Error creating threadn");
        }
    }

    for (int i = 0; i < num_threads; i++)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
        }
    }

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("multi device (pthreads) done after: %.3fms \n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    verify();
}

void run_single_kernel()
{
    hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N);
    hipEventRecord(start, 0);
    kernel<<<80, 1024>>>(0, N);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("single device done after: %.3fms \n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    verify();
}

void run_multi_kernel_new()
{
    checkCuda(hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStream_t streams[num_gpus];

    for (int i = 0; i < num_gpus; i++)
    {
        printf("\tcudaMemAdvise and prefetch for device %i\n", i);
        size_t perGpu = (N + num_gpus - 1) / num_gpus;
        size_t start = i * perGpu;
        size_t end = min(N, start + perGpu);
        hipSetDevice(i);
        hipStreamCreate(&streams[i]);
        hipMemAdvise(__memory__ + start, (end - start) * sizeof(uint), hipMemAdviseSetPreferredLocation, i);
        hipMemPrefetchAsync(__memory__ + start, (end - start) * sizeof(uint), i, streams[i]);
    }

    hipEventRecord(start, 0);

    for (int i = 0; i < num_gpus; i++)
    {
        size_t perGpu = (N + num_gpus - 1) / num_gpus;
        size_t start = i * perGpu;
        size_t end = min(N, start + perGpu);
        checkCuda(hipSetDevice(i));
        checkCuda(hipStreamCreate(&streams[i]));
        printf("\tstarting device %i on data [%lu, %lu) total: %lu\n", i, start, end, N);
        kernel<<<80, 1024, 0, streams[i]>>>(start, end);
    }

    for (int i = 0; i < num_gpus; i++)
    {
        hipStreamSynchronize(streams[i]);
    }

    hipSetDevice(0);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("multi device (new) done after: %.3fms \n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    verify();
}

int main()
{
    hipGetDeviceCount(&num_gpus);

    hipMallocManaged(&__memory__, sizeof(uint) * N);

    run_multi_kernel();

    run_single_kernel();

    run_multi_kernel_new();

    hipFree(__memory__);

    hipError_t result;
    result = hipGetLastError();
    if (hipSuccess != result)
    {
        fprintf(stderr, "error during execution: %s\n", hipGetErrorString(result));
    }

    return 0;
}