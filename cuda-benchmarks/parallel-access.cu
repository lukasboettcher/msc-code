#include "hip/hip_runtime.h"
#include <pthread.h>
#include <stdio.h>
#include <chrono>
#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char *const func, const char *const file, const int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error at: '%s:%i'\n\t%s %s\n", file, line, hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}

using myclock = std::chrono::high_resolution_clock;
myclock::time_point before, after;

int args[100];
// const size_t N = 1 * 1024 * 1024 * 1024L;
const size_t N = 1 * 1024 * 1024 * 1024L;
int num_gpus;

typedef size_t data_t;

__device__ __managed__ data_t *__memory__;
__managed__ data_t N_FIB = 100;

__device__ __host__ data_t fib(data_t n)
{
    data_t a = 0, b = 1, c, i;
    if (n == 0)
        return a;
    for (i = 2; i <= n; i++)
    {
        c = a + b;
        a = b;
        b = c;
    }
    return b;
}

__global__ void kernel(size_t start, size_t end)
{
    data_t tid = start + threadIdx.x + blockIdx.x * blockDim.x;
    for (data_t i = tid; i < end; i += blockDim.x * gridDim.x)
    {
        __memory__[i] = i + fib(N_FIB);
    }
}

void *launch_kernel(void *arg)
{
    int threadId = *((int *)arg);

    hipSetDevice(threadId);

    size_t perGpu = (N + num_gpus - 1) / num_gpus;

    size_t start = threadId * perGpu;
    size_t end = min(N, start + perGpu);

    hipMemAdvise(__memory__ + start, perGpu, hipMemAdviseSetPreferredLocation, threadId);
    hipMemPrefetchAsync(__memory__ + start, perGpu, threadId, hipStreamPerThread);

    printf("\tstarting thread %i w/ start: %lu end: %lu | total: %lu, pergpu: %lu\n", threadId, start, end, N, perGpu);

    kernel<<<80, 1024>>>(start, end);

    hipStreamSynchronize(hipStreamPerThread);

    return NULL;
}

void verify()
{
    data_t static_fib = fib(N_FIB);
    for (size_t i = 0; i < N; i++)
        // assert(__memory__[i] == i);
        if (__memory__[i] != i + static_fib)
        {
            fprintf(stderr, "error, w/ iter: %lu\n", i);
            break;
        }
}

void run_multi_kernel_threaded()
{
    hipMemset(__memory__, UCHAR_MAX, sizeof(data_t) * N);

    const int num_threads = num_gpus;

    pthread_t threads[num_threads];

    before = myclock::now();

    for (int i = 0; i < num_threads; i++)
    {
        args[i] = i;
        if (pthread_create(&threads[i], NULL, launch_kernel, &args[i]))
        {
            fprintf(stderr, "Error creating threadn");
        }
    }

    for (int i = 0; i < num_threads; i++)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
        }
    }

    after = myclock::now();
    printf("multi device (pthreads) done after: %.3fms \n", std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before).count());

    verify();
}

void run_single_kernel()
{
    hipMemset(__memory__, UCHAR_MAX, sizeof(data_t) * N);

    hipMemAdvise(__memory__, sizeof(data_t) * N, hipMemAdviseSetPreferredLocation, 0);
    hipMemPrefetchAsync(__memory__, sizeof(data_t) * N, 0, 0);
    before = myclock::now();
    kernel<<<80, 1024, 0, 0>>>(0, N);
    hipDeviceSynchronize();
    after = myclock::now();

    printf("single device done after: %.3fms \n", std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before).count());

    verify();
}

void run_multi_kernel()
{
    checkCuda(hipMemset(__memory__, UCHAR_MAX, sizeof(data_t) * N));

    hipStream_t streams[num_gpus];

#pragma omp parallel for num_threads(num_gpus)
    for (int i = 0; i < num_gpus; i++)
    {
        printf("\tcudaMemAdvise and prefetch for device %i\n", i);
        size_t perGpu = (N + num_gpus - 1) / num_gpus;
        size_t start = i * perGpu;
        size_t end = min(N, start + perGpu);
        hipSetDevice(i);
        hipFree(0);
        hipStreamCreate(&streams[i]);
        hipMemAdvise(__memory__ + start, (end - start) * sizeof(data_t), hipMemAdviseSetPreferredLocation, i);
        hipMemPrefetchAsync(__memory__ + start, (end - start) * sizeof(data_t), i, streams[i]);
    }

    before = myclock::now();

#pragma omp parallel for num_threads(num_gpus)
    for (int i = 0; i < num_gpus; i++)
    {
        size_t perGpu = (N + num_gpus - 1) / num_gpus;
        size_t start = i * perGpu;
        size_t end = min(N, start + perGpu);
        checkCuda(hipSetDevice(i));
        printf("\tstarting device %i on data [%lu, %lu) total: %lu\n", i, start, end, N);
        kernel<<<80, 1024, 0, streams[i]>>>(start, end);
    }

#pragma omp parallel for num_threads(num_gpus)
    for (int i = 0; i < num_gpus; i++)
    {
        checkCuda(hipStreamSynchronize(streams[i]));
    }

    after = myclock::now();

    printf("multi device (new) done after: %.3fms \n", std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before).count());

    for (int i = 0; i < num_gpus; i++)
    {
        hipSetDevice(i);
        checkCuda(hipStreamDestroy(streams[i]));
    }

    verify();
}

int main()
{
    hipGetDeviceCount(&num_gpus);

    hipMallocManaged(&__memory__, sizeof(data_t) * N);

    // run_multi_kernel_threaded();

    run_single_kernel();

    run_multi_kernel();

    hipFree(__memory__);

    hipError_t result;
    result = hipGetLastError();
    if (hipSuccess != result)
    {
        fprintf(stderr, "error during execution: %s\n", hipGetErrorString(result));
    }

    return 0;
}