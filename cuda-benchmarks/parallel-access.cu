#include "hip/hip_runtime.h"
#include <pthread.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <time.h>
#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char *const func, const char *const file, const int line)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "CUDA Runtime Error at: '%s:%i'\n\t%s %s\n", file, line, hipGetErrorString(err), func);
    exit(EXIT_FAILURE);
  }
}



int args[100];
// const size_t N = 1 * 1024 * 1024 * 1024L;
const size_t N = 128 * 1024 * 1024L;
int num_gpus;

__device__ __managed__ uint *__memory__;

__global__ void kernel(size_t start, size_t end, size_t n)
{
    uint tid = start + threadIdx.x + blockIdx.x * blockDim.x;
    for (uint i = tid; i < n && i < end; i += blockDim.x * gridDim.x)
    {
        __memory__[i] = i;
    }
}

void *launch_kernel(void *arg)
{
    int threadId = *((int *)arg);
    int threadsPerBlock = 1024;
    int blocksPerGrid = 80;

    hipSetDevice(threadId);

    size_t perGpu = (N + num_gpus - 1) / num_gpus;

    size_t start = threadId * perGpu;

    hipMemAdvise(__memory__ + start, perGpu, hipMemAdviseSetPreferredLocation, threadId);
    hipMemPrefetchAsync(__memory__ + start, perGpu, threadId, hipStreamPerThread);

    // printf("starting thread %i w/ start: %i end: %i | total: %i, pergpu: %i\n", threadId, start, start + perGpu, N, perGpu);

    kernel<<<blocksPerGrid, threadsPerBlock>>>(start, start + perGpu, N);

    hipStreamSynchronize(hipStreamPerThread);

    return NULL;
}

void verify()
{
    for (size_t i = 0; i < N; i++)
        // assert(__memory__[i] == i);
        if (__memory__[i] != i)
        {
            fprintf(stderr, "error, w/ iter: %lu\n", i);
            break;
        }
}

void run_multi_kernel()
{
    hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int num_threads = num_gpus;

    pthread_t threads[num_threads];

    hipEventRecord(start, 0);

    for (int i = 0; i < num_threads; i++)
    {
        args[i] = i;
        if (pthread_create(&threads[i], NULL, launch_kernel, &args[i]))
        {
            fprintf(stderr, "Error creating threadn");
        }
    }

    for (int i = 0; i < num_threads; i++)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
        }
    }

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("multi device done after: %.3fms \n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    verify();
}

void run_single_kernel()
{
    hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N);
    hipEventRecord(start, 0);
    int threadsPerBlock = 1024;
    int blocksPerGrid = 80;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(0, N, N);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("single device done after: %.3fms \n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    verify();
}

void run_multi_kernel_new()
{
    checkCuda(hipMemset(__memory__, UCHAR_MAX, sizeof(uint) * N));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStream_t streams[num_gpus];

    hipEventRecord(start, 0);

    for (int i = 0; i < num_gpus; i++)
    {
        size_t perGpu = (N + num_gpus - 1) / num_gpus;

        size_t start = i * perGpu;
        checkCuda(hipSetDevice(i));
        checkCuda(hipStreamCreate(&streams[i]));

        // hipMemAdvise(__memory__ + start, perGpu, hipMemAdviseSetPreferredLocation, i);
        // hipMemPrefetchAsync(__memory__ + start, perGpu, i, streams[i]);

        kernel<<<80, 1024, 0, streams[i]>>>(start, start + perGpu, N);
        // checkCuda(hipStreamSynchronize(streams[i]));
    }
    hipSetDevice(0);
    checkCuda(hipDeviceSynchronize());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("multi device (new) done after: %.3fms \n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    verify();
}

int main()
{
    hipGetDeviceCount(&num_gpus);

    hipMallocManaged(&__memory__, sizeof(uint) * N);

    // run_multi_kernel();

    run_single_kernel();

    run_multi_kernel_new();

    hipFree(__memory__);

    hipError_t result;
    result = hipGetLastError();
    if (hipSuccess != result){
        fprintf(stderr, "error during execution: %s\n", hipGetErrorString(result));
    }

    return 0;
}