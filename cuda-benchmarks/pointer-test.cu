
#include <hip/hip_runtime.h>
#include "stdio.h"

__managed__ int* x;

__global__ void kernel(int *a)
{
    a[0] = 10;
    x[0] = 100; 
}

int main(int argc, char const *argv[])
{
    int *host_ptr, *device_ptr;
    hipHostAlloc(&host_ptr, sizeof(int), 0 | 0); // cudaHostAllocMapped
    x = host_ptr;
    hipHostGetDevicePointer(&device_ptr, host_ptr, 0);
    kernel<<<1, 1>>>(host_ptr);
    hipDeviceSynchronize();

    printf("host ptr: %p\n", host_ptr);
    printf("device ptr: %p\n", device_ptr);
    printf("%i\n", x[0]);
    return 0;
}
