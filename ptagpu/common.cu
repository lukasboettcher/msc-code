#include "hip/hip_runtime.h"
#include "common.cuh"

/**
 * getHeadIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 *
 * \return index of the
 *
 */
__host__ __device__ size_t getHeadIndex(uint src, uint *graph)
{
    return 0;
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ __device__ void insertEdge(uint src, uint dst, uint *graph)
{
    uint index = 1 << 12;
    while (graph[index] != UINT_MAX)
        index += ELEMENT_WIDTH;
    for (size_t i = 0; i < ELEMENT_WIDTH; i++)
    {
        graph[index + i] = 0;
    }

    graph[index] |= 1 << dst;
    graph[src] = index;
}

__global__ void kernel(int n, uint *A, uint *B, uint *C)
{
}

__host__ int run()
{
    // CUDA kernel to add elements of two arrays

    int N = 1 << 20;
    uint *pts, *prevPtsDiff, *currPtsDiff, *invCopy, *invStore, *invLoad;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&prevPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&currPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invCopy, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invStore, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invLoad, N * sizeof(uint1)));

    // set all values to UINT_MAX
    hipMemset(pts, UINT_MAX, N);
    hipMemset(prevPtsDiff, UINT_MAX, N);
    hipMemset(currPtsDiff, UINT_MAX, N);
    hipMemset(invCopy, UINT_MAX, N);
    hipMemset(invStore, UINT_MAX, N);
    hipMemset(invLoad, UINT_MAX, N);

    insertEdge(1, 0, pts);
    insertEdge(2, 1, invLoad);
    insertEdge(1, 3, invStore);
    insertEdge(3, 4, pts);

    // Launch kernel on 1M elements on the GPU

    dim3 numBlocks(1);
    dim3 threadsPerBlock(warpSize, 1024 / warpSize);
    kernel<<<numBlocks, threadsPerBlock>>>(5, invCopy, pts, pts);

    // Wait for GPU to finish before accessing on host
    checkCuda(hipDeviceSynchronize());

    // Free memory
    checkCuda(hipFree(pts));
    checkCuda(hipFree(prevPtsDiff));
    checkCuda(hipFree(currPtsDiff));
    checkCuda(hipFree(invCopy));
    checkCuda(hipFree(invStore));
    checkCuda(hipFree(invLoad));

    return 0;
}