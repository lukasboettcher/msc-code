#include "hip/hip_runtime.h"
#include "common.cuh"

/**
 * __ptsFreeList__
 * this is the head of the free list
 * keeps track of last allocated memory location
 * access needs to be atomic to prevent collisions
 *
 */
__device__ __managed__ uint __freeList__[N_TYPES];

/**
 * __reservedHeader__
 *
 * this variable represents to max number of nodes
 *
 * is to be initialized with enough overhead
 * to allow adding further nodes via gep offsets calculations
 *
 */
__device__ __managed__ uint __reservedHeader__;

/**
 * flag that keeps track of remaining work
 * if true, no next iteration needed
 *
 */
__device__ __managed__ bool __done__ = true;

__device__ __managed__ uint *__key__;
__device__ __managed__ uint *__val__;
__device__ __managed__ uint *__keyAux__;

__device__ uint __numKeys__;

__device__ __managed__ uint __counter__ = 0;

__device__ uint __worklistIndex0__ = 0;
__device__ uint __worklistIndex1__ = 0;

__device__ uint __storeMapHead__ = 0;

/**
 * device pointers for the pts bitvectors
 * these need to be accesses adhoc
 * so are written to device symbols permanently
 *
 */
__device__ __managed__ uint *__memory__;

/**
 * getIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 * \param rel relation for which to get the head index
 *
 * \return index of the bitvector
 *
 */
__host__ __device__ size_t getIndex(uint src, uint rel)
{
    switch (rel)
    {
    case PTS:
        return OFFSET_PTS + (32 * src);
    case PTS_CURR:
        return OFFSET_PTS_CURR + (32 * src);
    case PTS_NEXT:
        return OFFSET_PTS_NEXT + (32 * src);
    case COPY:
        return OFFSET_COPY + (32 * src);
    case LOAD:
        return OFFSET_LOAD + (32 * src);
    case STORE:
        return OFFSET_STORE + (32 * src);
    }
    // uint index = __memory__[src * N_TYPES + rel];
    return src * 32;
}

__host__ uint incEdgeCouterHost(int type)
{
    uint index = __freeList__[type];
    __freeList__[type] += 32;
    return index;
}

__device__ uint incEdgeCouter(int type)
{
    __shared__ volatile uint _shared_[THREADS_PER_BLOCK / WARP_SIZE];
    if (threadIdx.x == 0)
    {
        _shared_[threadIdx.y] = atomicAdd(&__freeList__[type], 32);
    }
    return _shared_[threadIdx.y];
}

__device__ uint insertEdgeDevice(uint src, uint dst, uint *graph, uint toRel)
{
    uint index = getIndex(src, toRel);
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    uint myBits = 0;

    if (threadIdx.x == word)
        myBits = 1 << bit;
    else if (threadIdx.x == BASE)
        myBits = base;
    else if (threadIdx.x == NEXT)
        myBits = UINT_MAX;

    while (1)
    {
        uint toBits = graph[index + threadIdx.x];
        uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
        if (toBase == UINT_MAX)
        {
            graph[index + threadIdx.x] = myBits;
            return index;
        }
        if (toBase == base)
        {
            uint orBits = toBits | myBits;
            if (orBits != toBits && threadIdx.x < NEXT)
                graph[index + threadIdx.x] = orBits;

            return index;
        }
        if (toBase < base)
        {
            uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            if (toNext == UINT_MAX)
            {
                uint newIndex = incEdgeCouter(toRel);
                graph[index + NEXT] = newIndex;
                graph[newIndex + threadIdx.x] = myBits;
                return newIndex;
            }
            index = toNext;
        }
        else
        {
            uint newIndex = incEdgeCouter(toRel);
            graph[newIndex + threadIdx.x] = toBits;
            uint val = threadIdx.x == NEXT ? newIndex : myBits;
            graph[index + threadIdx.x] = val;
            return index;
        }
    }
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ void insertEdge(uint src, uint dst, uint *graph, uint toRel)
{
    uint index = getIndex(src, toRel);
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    // printf("inserting edge %u -> %u\n\tindex: %u\n\tbase: %u\n\tword: %u\n\tbit: %u\n", src, dst, index, base, word, bit);

    if (graph[index + BASE] == UINT_MAX)
    {
        for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
            graph[index + i] = 0;
        graph[index + BASE] = base;
        graph[index + word] |= 1 << bit;
        return;
    }

    while (1)
    {
        uint toBase = graph[index + BASE];
        uint toNext = graph[index + NEXT];

        if (toBase == UINT_MAX)
        {
            for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                graph[index + i] = 0;
            graph[index + BASE] = base;
            graph[index + word] |= 1 << bit;
            return;
        }
        if (toBase < base)
        {
            if (toNext == UINT_MAX)
            {
                uint nextIndex = incEdgeCouterHost(toRel);
                graph[index + NEXT] = nextIndex;

                for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                    graph[nextIndex + i] = 0;
                graph[nextIndex + BASE] = base;
                graph[nextIndex + word] |= 1 << bit;

                return;
            }

            index = toNext;
        }
        else if (base == toBase)
        {
            graph[index + word] |= 1 << bit;
            return;
        }
        else if (toBase > base)
        {

            uint nextIndex = incEdgeCouterHost(toRel);
            for (size_t i = 0; i < ELEMENT_WIDTH; i++)
                graph[nextIndex + i] = graph[index + i];
            for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                graph[nextIndex + i] = 0;
            graph[index + BASE] = base;
            graph[index + NEXT] = nextIndex;
            graph[index + word] |= 1 << bit;
        }
    }
}

template <uint fromRel, uint toRel>
__device__ void mergeBitvectors(const uint to, const uint numDstNodes, uint *_shared_);

template <uint fromRel, uint toRel>
__device__ void collectBitvectorTargets(const uint to, const uint bits, const uint base, uint *storage, uint &usedStorage);

__device__ inline uint getAndIncrement(uint *counter, uint delta)
{
    __shared__ volatile uint _shared_[THREADS_PER_BLOCK / WARP_SIZE];
    if (!threadIdx.x)
    {
        _shared_[threadIdx.y] = atomicAdd(counter, delta);
    }
    return _shared_[threadIdx.y];
}

__device__ void insert_store_map(const uint src, uint *const _shared_, uint numFrom)
{
    const uint storeIndex = getIndex(src, STORE);
    for (int i = 0; i < numFrom; i += 32)
    {
        uint size = min(numFrom - i, 32);
        uint next = getAndIncrement(&__storeMapHead__, size);
        // TODO: we need to make sure that (next + threadIdx.x < MAX_HASH_SIZE)
        if (threadIdx.x < size)
        {
            __key__[next + threadIdx.x] = _shared_[i + threadIdx.x]; // at most 2 transactions
            __val__[next + threadIdx.x] = src;
        }
    }
}

__device__ void mergeBitvectorCopy(uint to, uint fromIndex, uint *storage, const uint toRel)
{
    uint toIndex = getIndex(to, toRel);
    if (fromIndex == toIndex)
    {
        return;
    }
    // read dst out edges
    uint fromBits = __memory__[fromIndex + threadIdx.x];
    // get the base from thread nr 30
    uint fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
    // terminate if no data in from bitvector
    if (fromBase == UINT_MAX)
    {
        return;
    }
    // get the next index from thread nr 31
    uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);

    // share needed data for to indices
    uint toBits = __memory__[toIndex + threadIdx.x];
    uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
    uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);

    // keep count of used storage in shared memory
    // this storage is adjacent to previous collectBitvectorTargets memory
    uint numFrom = 0;
    while (1)
    {
        if (toBase == fromBase)
        {
            // union the bits, adding the new edges
            uint orBits = fromBits | toBits;
            uint diffs = __any_sync(0x7FFFFFFF, orBits != toBits);
            bool nextWasUndef = false;
            if (toNext == UINT_MAX && fromNext != UINT_MAX)
            {
                toNext = incEdgeCouter(toRel);
                nextWasUndef = true;
            }
            // each thread gets a value that will be written back to memory
            uint val = threadIdx.x == NEXT ? toNext : orBits;
            if (val != toBits)
                __memory__[toIndex + threadIdx.x] = val;

            // as we are merging into copy,
            // we need to also merge the underlying pts sets
            // we do this by running collectBitvectorTargets
            // and then merge thos pts edges again at the end of this loop
            if (diffs)
            {
                uint diffBits = fromBits & ~toBits;
                collectBitvectorTargets<PTS, PTS_NEXT>(to, diffBits, fromBase, storage, numFrom);
            }
            // if no more bitvectors in origin, end loop
            if (fromNext == UINT_MAX)
            {
                break;
            }
            // else load next bits
            // keep in mind that we do not use insertBitvector
            // since we need to also merge pts edges
            // instead make toBits undefined manually
            // handle this in toBase > fromBase
            toIndex = toNext;
            if (nextWasUndef)
            {
                toBits = UINT_MAX;
                toBase = UINT_MAX;
                toNext = UINT_MAX;
            }
            else
            {
                toBits = __memory__[toIndex + threadIdx.x];
                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            }
            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
            fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
        }
        else if (toBase < fromBase)
        {
            // if toNext is undefined, we need to allocate a new element
            // after that, we can simply insert the origin bitvector
            if (toNext == UINT_MAX)
            {
                uint newNext = incEdgeCouter(toRel);
                __memory__[toIndex + NEXT] = newNext;
                assert(toIndex != newNext);
                toIndex = newNext;
                toBits = UINT_MAX;
                toBase = UINT_MAX;
            }
            else
            {
                toIndex = toNext;
                toBits = __memory__[toNext + threadIdx.x];
                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            }
        }
        else if (toBase > fromBase)
        {
            // compared to mergeBitvectorPts
            // we need to handle the toBase == UINT_MAX case here
            uint newVal;
            if (toBase == UINT_MAX)
            {
                newVal = fromNext == UINT_MAX ? UINT_MAX : incEdgeCouter(toRel);
            }
            else
            {
                newVal = incEdgeCouter(toRel);
                // write the current bits from the target element to a new location
                __memory__[newVal + threadIdx.x] = toBits;
            }

            // overwrite the current bits with fromBits (insert before node)
            fromBits = threadIdx.x == NEXT ? newVal : fromBits;
            __memory__[toIndex + threadIdx.x] = fromBits;

            // collect pts edges
            collectBitvectorTargets<PTS, PTS_NEXT>(to, fromBits, fromBase, storage, numFrom);

            // if next from element is defined, update the bits
            // if not, break for this element
            if (fromNext == UINT_MAX)
            {
                break;
            }
            toIndex = newVal;

            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
            fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
        }
    }
    // merge collected pts edges
    if (numFrom)
    {
        mergeBitvectors<PTS, PTS_NEXT>(to, numFrom, storage);
    }
}

__device__ void insertBitvector(uint toIndex, uint fromBits, uint toRel)
{
    while (1)
    {
        // use warp intrinsics to get next index in from memory
        uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
        // check if a new bitvector is required
        // if that is the case, allocate a new index for a new element
        uint toNext = fromNext == UINT_MAX ? UINT_MAX : incEdgeCouter(toRel);
        // handle the special next entry, since we can not reuse the fromNext bits
        uint val = threadIdx.x == NEXT ? toNext : fromBits;
        // write new values to target memory location
        __memory__[toIndex + threadIdx.x] = val;

        // exit if no more elements in from bitvector
        if (fromNext == UINT_MAX)
            return;
        toIndex = toNext;
        fromBits = __memory__[fromNext + threadIdx.x];
    }
}

__device__ void mergeBitvectorPts(uint to, uint fromIndex, const uint toRel)
{
    uint toIndex = getIndex(to, toRel);
    // read dst out edges
    uint fromBits = __memory__[fromIndex + threadIdx.x];
    // get the base from thread nr 30
    uint fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
    // terminate if no data in from bitvector
    if (fromBase == UINT_MAX)
        return;
    // get the next index from thread nr 31
    uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);

    // share needed data for to indices
    uint toBits = __memory__[toIndex + threadIdx.x];
    uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
    uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);

    if (toBase == UINT_MAX)
    {
        insertBitvector(toIndex, fromBits, toRel);
        return;
    }

    while (1)
    {
        if (toBase == fromBase)
        {
            // if target next is undefined, create new edge for more edges
            uint newToNext = (toNext == UINT_MAX && fromNext != UINT_MAX) ? incEdgeCouter(toRel) : toNext;
            // union the bits, adding the new edges
            uint orBits = fromBits | toBits;
            // each thread gets a value that will be written back to memory
            uint val = threadIdx.x == NEXT ? newToNext : orBits;
            if (val != toBits)
                __memory__[toIndex + threadIdx.x] = val;

            // if no more bitvectors in origin, end loop
            if (fromNext == UINT_MAX)
                return;

            // else load next bits
            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
            fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
            if (toNext == UINT_MAX)
            {
                insertBitvector(toIndex, fromBits, toRel);
                return;
            }
            toIndex = newToNext;
            toBits = __memory__[toNext + threadIdx.x];
            toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
            toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
        }
        else if (toBase < fromBase)
        {
            // if toNext is undefined, we need to allocate a new element
            // after that, we can simply insert the origin bitvector
            if (toNext == UINT_MAX)
            {
                toNext = incEdgeCouter(toRel);
                insertBitvector(toNext, fromBits, toRel);
                return;
            }
            // if toNext is defined, load those to bits for the next iteration
            toIndex = toNext;
            toBits = __memory__[toNext + threadIdx.x];
            toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
            toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
        }
        else if (toBase > fromBase)
        {
            // if toBase is greater than frombase
            // we need to insert another bitvector element before toindex
            // and shift the current element back (ref. linked lists)
            uint newIndex = incEdgeCouter(toRel);
            // write the current bits from the target element to a new location
            __memory__[newIndex + threadIdx.x] = toBits;
            // then overwrite the current bits with fromBits (insert before node)
            uint val = threadIdx.x == NEXT ? newIndex : fromBits;
            __memory__[toIndex + threadIdx.x] = val;

            // if next from element is defined, update the bits
            // if not, break for this element
            if (fromNext == UINT_MAX)
                return;

            toIndex = newIndex;

            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
            fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
        }
    }
}

template <uint fromRel, uint toRel>
__device__ void mergeBitvectors(const uint to, const uint numDstNodes, uint *_shared_)
{
    // go through all dst nodes, and union the out edges of that node w/ src's out nodes
    for (size_t i = 0; i < numDstNodes; i++)
    {
        uint fromIndex = getIndex(_shared_[i], fromRel);

        if (toRel == COPY)
        {
            mergeBitvectorCopy(to, fromIndex, _shared_ + 128, toRel);
        }
        else
        {
            mergeBitvectorPts(to, fromIndex, toRel);
        }
    }
}

template <uint fromRel, uint toRel>
__device__ void collectBitvectorTargets(const uint to, const uint bits, const uint base, uint *storage, uint &usedStorage)
{
    // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
    uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
    const uint threadMask = 1 << threadIdx.x;
    const uint myMask = threadMask - 1;
    while (nonEmptyThreads)
    {
        // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
        int leastThread = __ffs(nonEmptyThreads) - 1;
        // remove lsb from nonEmptyThreads (iteration step)
        nonEmptyThreads &= (nonEmptyThreads - 1);
        // share current bits with all threads in warp
        uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);

        // use the base and the word of the current thread's bits to calculate the target dst id
        uint var = getDstNode(base, leastThread, threadIdx.x);
        // check if this thread is looking at a dst node
        // uint bitActive = (var != 1U) && (current_bits & threadMask);
        uint bitActive = (current_bits & threadMask);
        // count threads that are looking at dst nodes
        uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
        uint numDstNodes = __popc(threadsWithDstNode);
        if (usedStorage + numDstNodes > 128)
        {
            if (toRel == STORE)
                insert_store_map(to, storage, usedStorage);
            else
                mergeBitvectors<fromRel, toRel>(to, usedStorage, storage);
            usedStorage = 0;
        }
        // calculate pos in shared mem, by counting prev threads that had a dst node
        uint pos = usedStorage + __popc(threadsWithDstNode & myMask);
        if (bitActive)
        {
            storage[pos] = var;
        }
        usedStorage += numDstNodes;
    }
}

__global__ void kernel_store2copy(const uint n)
{
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 256];
    uint *const _shared_ = &_sh_[threadIdx.y * 256];
    for (uint i = blockIdx.x * blockDim.y + threadIdx.y; i < n - 1; i += blockDim.y * gridDim.x)
    {
        uint idx = __keyAux__[i];
        uint idx_next = __keyAux__[i + 1];

        // load the pts target, this should not change for the next totalDstNodes
        uint pts_target = __key__[idx];

        for (uint j = idx; j < idx_next; j += 32)
        {
            uint numDstNodes = min(idx_next - j, 32U);
            if (j + threadIdx.x < idx_next)
            {
                _shared_[threadIdx.x] = __val__[j + threadIdx.x];
            }
            mergeBitvectors<STORE, COPY>(pts_target, numDstNodes, _shared_);
        }
    }
}

__global__ void kernel_insert_edges(const uint n, const uint n_unique, uint *from, uint *to, uint *ofst, uint *memory, int rel)
{
    uint index = blockIdx.x * blockDim.y + threadIdx.y;
    uint stride = blockDim.y * gridDim.x;
    uint src, dst, offset, offset_next, j;
    for (int i = index; i < n_unique; i += stride)
    {

        offset = ofst[i];
        offset_next = i == (n_unique - 1) ? n : ofst[i + 1];
        src = from[offset];

        for (j = offset; j < offset_next; j++)
        {
            dst = to[j];
            insertEdgeDevice(src, dst, memory, rel);
        }
    }
}

__host__ void insertEdges(edgeSet *edges, uint *memory, int inv, int rel)
{
    uint *from, *to, *ofst, N;

    N = edges->second.size();

    checkCuda(hipMallocManaged(&from, N * sizeof(unsigned int)));
    checkCuda(hipMallocManaged(&to, N * sizeof(unsigned int)));
    checkCuda(hipMallocManaged(&ofst, N * sizeof(unsigned int)));

    if (inv)
    {
        memcpy(from, edges->second.data(), N * sizeof(unsigned int));
        memcpy(to, edges->first.data(), N * sizeof(unsigned int));
    }
    else
    {
        memcpy(from, edges->first.data(), N * sizeof(unsigned int));
        memcpy(to, edges->second.data(), N * sizeof(unsigned int));
    }

    thrust::sort_by_key(thrust::device, from, from + N, to);
    long numUnique = thrust::unique_by_key_copy(thrust::device, from, from + N, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), ofst).second - ofst;

    dim3 numBlocks(N_BLOCKS);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);

    checkCuda(hipDeviceSynchronize());
    kernel_insert_edges<<<numBlocks, threadsPerBlock>>>(N, numUnique, from, to, ofst, memory, rel);
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipFree(from));
    checkCuda(hipFree(to));
    checkCuda(hipFree(ofst));
}

/**
 * collect pts targets for src
 */
__host__ void collectFromBitvector(uint src, uint *memory, std::vector<uint> &pts)
{
    uint index = getIndex(src, PTS);
    uint base, next, bits, ptsTarget;

    while (index != UINT_MAX)
    {
        base = memory[index + BASE];
        next = memory[index + NEXT];

        if (base == UINT_MAX)
        {
            break;
        }

        for (size_t j = 0; j < BASE; j++)
        {
            bits = memory[index + j];
            for (size_t k = 0; k < 32; k++)
            {
                if (1 << k & bits)
                {
                    // calculate target from bitvector
                    ptsTarget = 960 * base + 32 * j + k;
                    pts.push_back(ptsTarget);
                }
            }
        }
        index = next;
    }
}

__host__ uint handleGepEdges(uint *memory, void *consG, void *pag)
{
    edgeSet newPts;
    handleGepsSVF(consG, pag, memory, newPts);
    insertEdges(&newPts, memory, 0, PTS_NEXT);
    uint nodeCount = getNodeCount(consG);
    return nodeCount;
}

__host__ bool aliasBV(uint a, uint b, uint *memory)
{
    std::vector<uint> ptsA, ptsB;

    collectFromBitvector(a, memory, ptsA);
    collectFromBitvector(b, memory, ptsB);

    for (uint target : ptsA)
        if (std::find(ptsB.begin(), ptsB.end(), target) != ptsB.end())
            return true;

    return false;
}

__device__ void cloneAndLink(uint var, const uint ptsIndex, uint &currDiffPtsIndex, const uint diffPtsBits, const uint diffPtsNext)
{
    insertBitvector(ptsIndex, diffPtsBits, PTS);
    if (currDiffPtsIndex != UINT_MAX)
    {
        __memory__[currDiffPtsIndex + NEXT] = ptsIndex;
        assert(currDiffPtsIndex != ptsIndex);
    }
    else
    {
        currDiffPtsIndex = getIndex(var, PTS_CURR);
        uint ptsBits = __memory__[ptsIndex + threadIdx.x];
        __memory__[currDiffPtsIndex + threadIdx.x] = ptsBits;
    }
}

/**
 * Update the current, next and total PTS sets of a variable. In the last iteration of the main
 * loop, points-to edges have been added to NEXT_DIFF_PTS. However, many of them might already be
 * present in PTS. The purpose of this function is to update PTS as PTS U NEXT_DIFF_PTS, and set
 * CURR_DIFF_PTS as the difference between the old and new PTS for the given variable.
 *
 * @param var ID of the variable
 * @param pts memory for all points to bitvectors
 * @param curr_pts memory for previous iterations points to bitvectors in working set
 * @param next_pts memory for all newly added pts bitvectors
 * @return true if new pts edges have been added to this variable
 */
__device__ bool updatePtsAndDiffPts(const uint var)
{
    // next next index
    const uint diffPtsHeadIndex = getIndex(var, PTS_NEXT);

    uint diffPtsBits = __memory__[diffPtsHeadIndex + threadIdx.x];
    uint diffPtsBase = __shfl_sync(0xFFFFFFFF, diffPtsBits, 30);

    if (diffPtsBase == UINT_MAX)
    {
        return false;
    }

    uint diffPtsNext = __shfl_sync(0xFFFFFFFF, diffPtsBits, 31);
    __memory__[diffPtsHeadIndex + threadIdx.x] = UINT_MAX;

    uint ptsIndex = getIndex(var, PTS);
    uint ptsBits = __memory__[ptsIndex + threadIdx.x];
    uint ptsBase = __shfl_sync(0xFFFFFFFF, ptsBits, 30);

    if (ptsBase == UINT_MAX)
    {
        // we pass ptsBase instead of UINT_MAX because it's also UINT_MAX but it can be modified
        cloneAndLink(var, ptsIndex, ptsBase, diffPtsBits, diffPtsNext);
        return true;
    }
    uint ptsNext = __shfl_sync(0xFFFFFFFF, ptsBits, 31);
    uint currDiffPtsIndex = UINT_MAX;
    while (1)
    {
        if (ptsBase > diffPtsBase)
        {
            uint newIndex = incEdgeCouter(PTS);
            __memory__[newIndex + threadIdx.x] = ptsBits;
            uint val = threadIdx.x == NEXT ? newIndex : diffPtsBits;
            __memory__[ptsIndex + threadIdx.x] = val;

            ptsIndex = newIndex;
            // update CURR_DIFF_PTS
            newIndex = currDiffPtsIndex == UINT_MAX ? getIndex(var, PTS_CURR) : incEdgeCouter(PTS_CURR);
            val = threadIdx.x == NEXT ? UINT_MAX : diffPtsBits;
            __memory__[newIndex + threadIdx.x] = val;
            if (currDiffPtsIndex != UINT_MAX)
            {
                __memory__[currDiffPtsIndex + NEXT] = newIndex;
                assert(currDiffPtsIndex != newIndex);
            }
            if (diffPtsNext == UINT_MAX)
            {
                return true;
            }
            currDiffPtsIndex = newIndex;

            diffPtsBits = __memory__[diffPtsNext + threadIdx.x];
            diffPtsBase = __shfl_sync(0xFFFFFFFF, diffPtsBits, 30);
            diffPtsNext = __shfl_sync(0xFFFFFFFF, diffPtsBits, 31);
        }
        else if (ptsBase == diffPtsBase)
        {
            uint newPtsNext = (ptsNext == UINT_MAX && diffPtsNext != UINT_MAX) ? incEdgeCouter(PTS) : ptsNext;
            uint orBits = threadIdx.x == NEXT ? newPtsNext : ptsBits | diffPtsBits;
            uint ballot = __ballot_sync(0xFFFFFFFF, orBits != ptsBits);
            if (ballot)
            {
                __memory__[ptsIndex + threadIdx.x] = orBits;
                if (ballot & ((1 << 30) - 1))
                {
                    // update CURR_DIFF_PTS
                    orBits = diffPtsBits & ~ptsBits;
                    if (threadIdx.x == BASE)
                    {
                        orBits = ptsBase;
                    }
                    else if (threadIdx.x == NEXT)
                    {
                        orBits = UINT_MAX;
                    }
                    uint newIndex;
                    if (currDiffPtsIndex != UINT_MAX)
                    {

                        newIndex = incEdgeCouter(PTS_CURR);
                        __memory__[currDiffPtsIndex + NEXT] = newIndex;
                        assert(currDiffPtsIndex != newIndex);
                    }
                    else
                    {
                        newIndex = getIndex(var, PTS_CURR);
                    }
                    __memory__[newIndex + threadIdx.x] = orBits;
                    currDiffPtsIndex = newIndex;
                }
            }
            if (diffPtsNext == UINT_MAX)
            {
                return (currDiffPtsIndex != UINT_MAX);
            }
            diffPtsBits = __memory__[diffPtsNext + threadIdx.x];
            diffPtsBase = __shfl_sync(0xFFFFFFFF, diffPtsBits, 30);
            diffPtsNext = __shfl_sync(0xFFFFFFFF, diffPtsBits, 31);

            if (ptsNext == UINT_MAX)
            {
                cloneAndLink(var, newPtsNext, currDiffPtsIndex, diffPtsBits, diffPtsNext);
                return true;
            }
            ptsIndex = ptsNext;

            ptsBits = __memory__[ptsIndex + threadIdx.x];
            ptsBase = __shfl_sync(0xFFFFFFFF, ptsBits, 30);
            ptsNext = __shfl_sync(0xFFFFFFFF, ptsBits, 31);
        }
        else
        { // ptsBase < diffPtsBase
            if (ptsNext == UINT_MAX)
            {
                uint newPtsIndex = incEdgeCouter(PTS);
                __memory__[ptsIndex + NEXT] = newPtsIndex;
                assert(ptsIndex != newPtsIndex);
                cloneAndLink(var, newPtsIndex, currDiffPtsIndex, diffPtsBits, diffPtsNext);
                return true;
            }
            ptsIndex = ptsNext;
            ptsBits = __memory__[ptsIndex + threadIdx.x];
            ptsBase = __shfl_sync(0xFFFFFFFF, ptsBits, 30);
            ptsNext = __shfl_sync(0xFFFFFFFF, ptsBits, 31);
        }
    }
}

__global__ void kernel_memoryCheck(const uint n)
{
    __syncthreads();

    uint start = blockIdx.x * blockDim.y + threadIdx.y;
    uint stride = blockDim.y * gridDim.x;
    uint bits, base, next, index;

    for (int i = start; i < n; i += stride)
    {
        index = getIndex(i, PTS_CURR);
        while (index != UINT_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;

            next = __shfl_sync(0xFFFFFFFF, bits, 31);
            if (!threadIdx.x && next == index)
            {
                printf("huh?? currpts index: %u has smaller next: %u, freeList: %u\n", index, next, __freeList__[PTS_CURR]);
                break;
            }
            index = next;
        }
    }
    __syncthreads();

    for (int i = start; i < n; i += stride)
    {
        index = getIndex(i, PTS);
        while (index != UINT_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;

            next = __shfl_sync(0xFFFFFFFF, bits, 31);
            if (!threadIdx.x && next == index)
            {
                printf("huh?? pts index: %u has smaller next: %u, freeList: %u\n", index, next, __freeList__[PTS]);
                break;
            }
            index = next;
        }
    }
    __syncthreads();

    for (int i = start; i < n; i += stride)
    {
        index = getIndex(i, PTS_NEXT);
        while (index != UINT_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;

            next = __shfl_sync(0xFFFFFFFF, bits, 31);
            if (!threadIdx.x && next == index)
            {
                printf("huh?? nextpts index: %u has smaller next: %u, freeList: %u\n", index, next, __freeList__[PTS_NEXT]);
                break;
            }
            index = next;
        }
    }
    __syncthreads();
}

__global__ void kernel_count_pts(const uint n, uint rel)
{
    uint start = blockIdx.x * blockDim.y + threadIdx.y;
    uint stride = blockDim.y * gridDim.x;
    uint bits, base, next, index;

    for (int i = start; i < n; i += stride)
    {
        index = getIndex(i, rel);
        while (index != UINT_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;

            uint value = threadIdx.x < BASE ? __popc(bits) : 0;

            for (int i = 16; i >= 1; i /= 2)
                value += __shfl_xor_sync(0x3FFFFFFF, value, i);

            if (!threadIdx.x)
            {
                atomicAdd(&__counter__, value);
            }

            next = __shfl_sync(0xFFFFFFFF, bits, 31);

            index = next;
        }
    }
}

__device__ inline uint resetWorklistIndex()
{
    __syncthreads();
    uint numBlocks = gridDim.x;
    if (!((32 * threadIdx.y) + threadIdx.x) && atomicInc(&__counter__, numBlocks - 1) == (numBlocks - 1))
    {
        __worklistIndex0__ = 0;
        __counter__ = 0;
        return 1;
    }
    return 0;
}

__global__ void kernel_updatePts(const uint n)
{
    __done__ = true;
    bool newWork = false;
    for (uint i = blockIdx.x * blockDim.y + threadIdx.y; i < n; i += blockDim.y * gridDim.x)
    {
        bool newStuff = updatePtsAndDiffPts(i);
        newWork |= newStuff;
        if (!newStuff)
        {
            const uint currPtsHeadIndex = getIndex(i, PTS_CURR);
            __memory__[currPtsHeadIndex + threadIdx.x] = UINT_MAX;
        }
    }
    if (newWork)
    {
        __done__ = false;
    }
    if (resetWorklistIndex())
    {
        __freeList__[PTS_CURR] = n * 32;
        __freeList__[PTS_NEXT] = n * 32;
    }
}

template <uint originRel, uint fromRel, uint toRel>
__device__ void rewriteRule(const uint src, uint *const _shared_)
{
    uint usedShared = 0;
    uint index = getIndex(src, originRel);
    do
    {
        uint bits = __memory__[index + threadIdx.x];
        uint base = __shfl_sync(0xFFFFFFFF, bits, 30);
        if (base == UINT_MAX)
            break;

        collectBitvectorTargets<fromRel, toRel>(src, bits, base, _shared_, usedShared);
        index = __shfl_sync(0xFFFFFFFF, bits, 31);
    } while (index != UINT_MAX);
    if (usedShared)
    {
        if (fromRel == STORE)
            insert_store_map(src, _shared_, usedShared);
        else
            mergeBitvectors<fromRel, toRel>(src, usedShared, _shared_);
    }
}

__global__ void kernel(const uint n, const uint n_stores, uint *storeConstraints)
{
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 256];
    uint *const _shared_ = &_sh_[threadIdx.y * 256];
    uint to = n;
    uint src = getAndIncrement(&__worklistIndex1__, 1);
    while (src < to)
    {
        rewriteRule<COPY, PTS_CURR, PTS_NEXT>(src, _shared_ + 128);
        rewriteRule<LOAD, PTS_CURR, COPY>(src, _shared_);

        src = getAndIncrement(&__worklistIndex1__, 1);
    }
    to = n_stores;
    src = getAndIncrement(&__worklistIndex0__, 1);
    while (src < to)
    {
        src = storeConstraints[src];
        if (src != UINT_MAX)
        {
            rewriteRule<PTS_CURR, STORE, STORE>(src, _shared_);
        }
        src = getAndIncrement(&__worklistIndex0__, 1);
    }
    if (resetWorklistIndex())
    {
        __numKeys__ = __storeMapHead__ + 1;
        __storeMapHead__ = 0;
        __worklistIndex1__ = 0;
    }
}

__host__ void printWord(uint *memory, uint src, uint rel, bool isNodeId = true)
{
    // if (isNodeId)
    //     start *= 32;
    uint start;
    if (isNodeId)
        start = getIndex(src, rel);
    else
        start = src;

    for (size_t i = 0; i < 32; i++)
    {
        uint checkpoint = memory[start + i];
        std::cout << checkpoint << "\n";
        std::bitset<sizeof(uint) * 8> x(checkpoint);
        std::cout << x << '\n';
    }
}

__host__ uint *run(unsigned int numNodes, edgeSet *addrEdges, edgeSet *directEdges, edgeSet *loadEdges, edgeSet *storeEdges, void *consG, void *pag)
{
    setlocale(LC_NUMERIC, "");
    // hipDeviceProp_t prop; // CUDA device properties variable
    // checkCuda(hipGetDeviceProperties(&prop, 0));
    // printf("total global memory available:\n\t\t%lu\n", prop.totalGlobalMem);
    // printf("total bytes: \t%lu\n", SIZE_TOTAL_BYTES);
    int N = 1 << 20;
    size_t numStoreDst = storeEdges->second.size();
    uint *store_map_pts, *store_map_src, *store_map_idx, *storeConstraints, *memory;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&memory, SIZE_TOTAL_BYTES));
    checkCuda(hipMallocManaged(&store_map_pts, N * sizeof(uint)));
    checkCuda(hipMallocManaged(&store_map_src, N * sizeof(uint)));
    checkCuda(hipMallocManaged(&store_map_idx, N * sizeof(uint)));
    checkCuda(hipMallocManaged(&storeConstraints, numStoreDst * sizeof(uint)));

    // set all values to UINT_MAX
    hipMemset(memory, UCHAR_MAX, SIZE_TOTAL_BYTES);
    hipMemset(store_map_pts, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_src, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_idx, UCHAR_MAX, N * sizeof(unsigned int));

    // move the store constraints into managed memory and sort / unique
    memcpy(storeConstraints, storeEdges->second.data(), numStoreDst * sizeof(uint));
    thrust::sort(storeConstraints, storeConstraints + numStoreDst);
    size_t numStoreConstraints = thrust::unique(storeConstraints, storeConstraints + numStoreDst) - storeConstraints;

    // num of vertices
    size_t V{numNodes};

    // move managed memory ptrs into device memory
    __memory__ = memory;
    __key__ = store_map_pts;
    __val__ = store_map_src;
    __keyAux__ = store_map_idx;

    // reserve 20% for new edges added by gep offsets
    uint initNum = std::ceil(2 * V) * ELEMENT_WIDTH;
    uint freeList[N_TYPES] = {initNum, initNum, initNum, initNum, initNum, initNum};
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(__freeList__), freeList, N_TYPES * sizeof(uint)));


    insertEdges(addrEdges, memory, 1, PTS_NEXT);
    insertEdges(directEdges, memory, 1, COPY);
    insertEdges(loadEdges, memory, 1, LOAD);
    insertEdges(storeEdges, memory, 1, STORE);

    dim3 numBlocks(N_BLOCKS);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);

    while (1)
    {
        kernel_updatePts<<<numBlocks, threadsPerBlock>>>(V);
        checkCuda(hipDeviceSynchronize());


        if (__done__)
        {
            break;
        }

        kernel<<<numBlocks, threadsPerBlock>>>(V, numStoreConstraints, storeConstraints);
        checkCuda(hipDeviceSynchronize());

        thrust::sort_by_key(thrust::device, store_map_pts, store_map_pts + N, store_map_src);
        auto numSrcs = thrust::unique_by_key_copy(thrust::device, store_map_pts, store_map_pts + N, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), store_map_idx).second - store_map_idx;

        kernel_store2copy<<<numBlocks, threadsPerBlock>>>(numSrcs);
        checkCuda(hipDeviceSynchronize());
        uint Vnew = handleGepEdges(memory, consG, pag);
        V = Vnew;
    }
    // Free memory
    checkCuda(hipFree(store_map_pts));
    checkCuda(hipFree(store_map_src));
    checkCuda(hipFree(store_map_idx));

    return memory;
}