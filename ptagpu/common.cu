#include "hip/hip_runtime.h"
#include "common.cuh"

std::map<unsigned int *, unsigned int> testMap;

/**
 * __ptsFreeList__
 * this is the head of the free list
 * keeps track of last allocated memory location
 * access needs to be atomic to prevent collisions
 *
 */
__device__ uint __freeList__[N_TYPES];

/**
 * flag that keeps track of remaining work
 * if true, no next iteration needed
 *
 */
__device__ bool __done__ = true;

/**
 * device pointers for the pts bitvectors
 * these need to be accesses adhoc
 * so are written to device symbols permanently
 *
 */
__device__ uint *__pts__;
__device__ uint *__ptsCurr__;
__device__ uint *__ptsNext__;

/**
 * getHeadIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 *
 * \return index of the
 *
 */
__host__ __device__ size_t getHeadIndex(uint src, uint *graph)
{
    return 0;
}

__device__ uint incEdgeCouter(int type)
{
    __shared__ volatile uint _shared_[THREADS_PER_BLOCK / WARP_SIZE];
    if (threadIdx.x == 0)
    {
        _shared_[threadIdx.y] = atomicAdd(&__freeList__[type], 32);
    }
    return _shared_[threadIdx.y];
}

__device__ uint insertEdgeDevice(uint src, uint dst, uint *graph, uint toRel)
{
    uint index = src * 32;
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    uint myBits = 0;

    if (threadIdx.x == word)
        myBits = 1 << bit;
    else if (threadIdx.x == BASE)
        myBits = base;
    else if (threadIdx.x == NEXT)
        myBits = UINT_MAX;

    while (1)
    {
        uint toBits = graph[index + threadIdx.x];
        uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
        if (toBase == UINT_MAX)
        {
            graph[index + threadIdx.x] = myBits;
            return index;
        }
        if (toBase == base)
        {
            uint orBits = toBits | myBits;
            if (orBits != toBits && threadIdx.x < NEXT)
                graph[index + threadIdx.x] = orBits;

            return index;
        }
        if (toBase < base)
        {
            uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            if (toNext == UINT_MAX)
            {
                uint newIndex = incEdgeCouter(toRel);
                graph[index + NEXT] = newIndex;
                graph[newIndex + threadIdx.x] = myBits;
                return newIndex;
            }
            index = toNext;
        }
        else
        {
            uint newIndex = incEdgeCouter(toRel);
            graph[newIndex + threadIdx.x] = toBits;
            uint val = threadIdx.x == NEXT ? newIndex : myBits;
            graph[index + threadIdx.x] = val;
            return index;
        }
    }
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ void insertEdge(uint src, uint dst, uint *graph)
{
    uint index = src * 32;
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    // printf("inserting edge %u -> %u\n\tindex: %u\n\tbase: %u\n\tword: %u\n\tbit: %u\n", src, dst, index, base, word, bit);

    if (graph[index + BASE] == UINT_MAX)
    {
        for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
            graph[index + i] = 0;
        graph[index + BASE] = base;
        graph[index + word] |= 1 << bit;
        return;
    }

    while (1)
    {
        uint toBase = graph[index + BASE];
        uint toNext = graph[index + NEXT];

        if (toBase == UINT_MAX)
        {
            for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                graph[index + i] = 0;
            graph[index + BASE] = base;
            graph[index + word] |= 1 << bit;
            return;
        }
        if (toBase < base)
        {
            if (toNext == UINT_MAX)
            {
                uint nextIndex = ++testMap[graph] * 32;
                graph[index + NEXT] = nextIndex;

                for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                    graph[nextIndex + i] = 0;
                graph[nextIndex + BASE] = base;
                graph[nextIndex + word] |= 1 << bit;

                return;
            }

            index = toNext;
        }
        else if (base == toBase)
        {
            graph[index + word] |= 1 << bit;
            return;
        }
        else if (toBase > base)
        {

            uint nextIndex = ++testMap[graph] * 32;
            for (size_t i = 0; i < ELEMENT_WIDTH; i++)
                graph[nextIndex + i] = graph[index + i];
            for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                graph[nextIndex + i] = 0;
            graph[index + BASE] = base;
            graph[index + NEXT] = nextIndex;
            graph[index + word] |= 1 << bit;
        }
    }
}

__device__ void collectBitvectorTargets(const uint index, const uint bits, const uint base, uint *storage, uint &usedStorage, uint *originMemory, uint *targetMemory, const uint toRel);

__device__ void insertBitvector(const uint index, const uint *originMemory, uint *targetMemory, uint *storage, uint &usedStorage, uint toIndex, uint fromBits, uint toRel)
{
    while (1)
    {
        // use warp intrinsics to get next index in from memory
        uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
        // check if a new bitvector is required
        // if that is the case, allocate a new index for a new element
        uint toNext = fromNext == UINT_MAX ? UINT_MAX : incEdgeCouter(toRel);
        // handle the special next entry, since we can not reuse the fromNext bits
        uint val = threadIdx.x == NEXT ? toNext : fromBits;
        // write new values to target memory location
        targetMemory[toIndex + threadIdx.x] = val;

        if (toRel == COPY)
        {
            uint fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
            collectBitvectorTargets(index, fromBits, fromBase, storage + 128, usedStorage, __pts__, __ptsNext__, PTS_NEXT);
        }

        // exit if no more elements in from bitvector
        if (fromNext == UINT_MAX)
            return;
        toIndex = toNext;
        fromBits = originMemory[fromNext + threadIdx.x];
    }
}

__device__ void mergeBitvectors(const uint *origin, uint *target, const uint index, const uint numDstNodes, uint *_shared_, const uint toRel)
{
    // go through all dst nodes, and union the out edges of that node w/ src's out nodes
    for (size_t i = 0; i < numDstNodes; i++)
    {
        uint fromIndex = _shared_[i] * 32;
        // read dst out edges
        uint fromBits = origin[fromIndex + threadIdx.x];
        // get the base from thread nr 30
        uint fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
        // terminate if no data in from from bitvector
        if (fromBase == UINT_MAX)
            continue;
        // get the next index from thread nr 31
        uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);

        // share needed data for to indices
        uint toIndex = index;
        uint toBits = target[toIndex + threadIdx.x];
        uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
        uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);

        uint runloop = 1;

        if (toBase == UINT_MAX)
        {
            insertBitvector(origin, target, toIndex, fromBits, toRel);
            runloop = 0;
        }

        while (runloop)
        {
            if (toBase == fromBase)
            {
                // if target next is undefined, create new edge for more edges
                uint newToNext = (toNext == UINT_MAX && fromNext != UINT_MAX) ? incEdgeCouter(toRel) : toNext;
                // union the bits, adding the new edges
                uint orBits = fromBits | toBits;
                // each thread gets a value that will be written back to memory
                uint val = threadIdx.x == NEXT ? newToNext : orBits;
                if (val != toBits)
                {
                    target[toIndex + threadIdx.x] = val;
                }

                // if no more bitvectors in origin, end loop
                if (fromNext == UINT_MAX)
                {
                    break;
                }
                // else load next bits
                fromBits = origin[fromNext + threadIdx.x];
                fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
                fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
                if (toNext == UINT_MAX)
                {
                    insertBitvector(origin, target, toIndex, fromBits, toRel);
                    break;
                }
                toIndex = newToNext;
                toBits = target[toNext + threadIdx.x];
                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            }
            else if (toBase < fromBase)
            {
                // if toNext is undefined, we need to allocate a new element
                // after that, we can simply insert teh origin bitvector
                if (toNext == UINT_MAX)
                {
                    toNext = incEdgeCouter(toRel);
                    insertBitvector(origin, target, toNext, fromBits, toRel);
                    break;
                }
                // if toNext is defined, load those to bits for the next iteration
                toIndex = toNext;
                toBits = target[toNext + threadIdx.x];
                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            }
            else if (toBase > fromBase)
            {
                // if toBase is greater than frombase
                // we need to insert another bitvector element before toindex
                // and shift the current element back (ref. linked lists)
                uint newIndex = incEdgeCouter(toRel);
                // write the current bits from the target element to a new location
                target[newIndex + threadIdx.x] = toBits;
                // then overwrite the current bits with fromBits (insert before node)
                uint val = threadIdx.x == NEXT ? newIndex : fromBits;
                target[toIndex + threadIdx.x] = val;

                // if next from element is defined, update the bits
                // if not, break for this element
                if (fromNext == UINT_MAX)
                    break;

                toIndex = newIndex;

                fromBits = origin[fromNext + threadIdx.x];
                fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
                fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
            }
        }
    }
}

__device__ void collectBitvectorTargets(const uint index, const uint bits, const uint base, uint *storage, uint &usedStorage, uint *originMemory, uint *targetMemory, const uint toRel)
{
    // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
    uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
    const uint threadMask = 1 << threadIdx.x;
    const uint myMask = threadMask - 1;
    while (nonEmptyThreads)
    {
        // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
        int leastThread = __ffs(nonEmptyThreads) - 1;
        // remove lsb from nonEmptyThreads (iteration step)
        nonEmptyThreads &= (nonEmptyThreads - 1);
        // share current bits with all threads in warp
        uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);

        // use the base and the word of the current thread's bits to calculate the target dst id
        uint var = base * 30 * 32 + 32 * leastThread + threadIdx.x;
        // check if this thread is looking at a dst node
        // uint bitActive = (var != 1U) && (current_bits & threadMask);
        uint bitActive = (current_bits & threadMask);
        // count threads that are looking at dst nodes
        uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
        uint numDstNodes = __popc(threadsWithDstNode);
        if (usedStorage + numDstNodes > 128)
        {
            // insert_store_map(index, usedStorage, storage, originMemory, targetMemory);
            mergeBitvectors(originMemory, targetMemory, index, numDstNodes, storage, toRel);
            usedStorage = 0;
        }
        // calculate pos in shared mem, by counting prev threads that had a dst node
        uint pos = usedStorage + __popc(threadsWithDstNode & myMask);
        if (bitActive)
        {
            storage[pos] = var;
        }
        usedStorage += numDstNodes;
    }
}

__global__ void kernel(int n, uint *A, uint *B, uint *C, uint toRel)
{
    // each warp gets a shared block for one access to global memory
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 256];
    uint *const _shared_ = &_sh_[threadIdx.y * 256];
    uint usedShared = 0;
    for (uint src = blockIdx.x * blockDim.y + threadIdx.y; src < n; src += blockDim.y * gridDim.x)
    {
        uint index = src * 32;
        do
        {
            uint bits = A[index + threadIdx.x];
            uint base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;

            collectBitvectorTargets(src * 32, bits, base, _shared_, usedShared, B, C, toRel);
            index = __shfl_sync(0xFFFFFFFF, bits, 31);
        } while (index != UINT_MAX);
        if (usedShared)
        {
            mergeBitvectors(B, C, src * 32, usedShared, _shared_, toRel);
        }
    }
}

__device__ uint store_map_head = 0;

__device__ void insert_store_map(const uint src, const uint n, uint *const list, uint *store_map_pts, uint *store_map_src)
{
    for (int i = 0; i < n; i += 32)
    {
        uint size = min(n - i, 32);
        uint next;
        if (!threadIdx.x)
        {
            next = atomicAdd(&store_map_head, size);
        }
        next = __shfl_sync(0xFFFFFFFF, next, 0);
        if (threadIdx.x < size)
        {
            store_map_pts[next + threadIdx.x] = list[i + threadIdx.x];
            // store_map_src[next + threadIdx.x] = src_index;
            store_map_src[next + threadIdx.x] = src;
        }
    }
}

/**
 * Kernel for store edges,
 * here we need to collect all store edges that share a pts edge
 * and selectively assign them to the same warps
 * so that we save on synchronization between warps
 */
__global__ void kernel_store(int n, uint *A, uint *B, uint *C)
{
    // each warp gets a shared block for one access to global memory
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 128];
    uint *const _shared_ = &_sh_[threadIdx.y * 128];
    for (uint src = blockIdx.x * blockDim.y + threadIdx.y; src < n; src += blockDim.y * gridDim.x)
    {
        uint index = src * 32;
        uint usedShared = 0;
        do
        {
            uint bits = A[index + threadIdx.x];
            uint base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;
            // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
            uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
            const uint threadMask = 1 << threadIdx.x;
            const uint myMask = threadMask - 1;
            while (nonEmptyThreads)
            {
                // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
                int leastThread = __ffs(nonEmptyThreads) - 1;
                // remove lsb from nonEmptyThreads (iteration step)
                nonEmptyThreads &= (nonEmptyThreads - 1);
                // share current bits with all threads in warp
                uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);

                // use the base and the word of the current thread's bits to calculate the target dst id
                uint var = base * 30 * 32 + 32 * leastThread + threadIdx.x;
                // check if this thread is looking at a dst node
                // uint bitActive = (var != 1U) && (current_bits & threadMask);
                uint bitActive = (current_bits & threadMask);
                // count threads that are looking at dst nodes
                uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
                uint numDstNodes = __popc(threadsWithDstNode);
                if (usedShared + numDstNodes > 128)
                {
                    insert_store_map(index, usedShared, _shared_, B, C);
                    usedShared = 0;
                }
                // calculate pos in shared mem, by counting prev threads that had a dst node
                uint pos = usedShared + __popc(threadsWithDstNode & myMask);
                if (bitActive)
                {
                    _shared_[pos] = var;
                }
                usedShared += numDstNodes;
            }
            index = __shfl_sync(0xFFFFFFFF, bits, 31);
        } while (index != UINT_MAX);
        if (usedShared)
        {
            insert_store_map(src, usedShared, _shared_, B, C);
        }
    }
}

__global__ void kernel_store2copy(const uint n, uint *store_map_pts, uint *store_map_src, uint *store_map_idx, uint *pts, uint *store, uint *invCopy, uint toRel)
{
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 256];
    uint *const _shared_ = &_sh_[threadIdx.y * 256];
    for (uint i = blockIdx.x * blockDim.y + threadIdx.y; i < n - 1; i += blockDim.y * gridDim.x)
    {
        uint idx = store_map_idx[i];
        uint idx_next = store_map_idx[i + 1];

        // load the pts target, this should no change for the next totalDstNodes
        uint pts_target = store_map_pts[idx];

        for (uint j = idx; j < idx_next; j += 32)
        {
            uint numDstNodes = min(idx_next - j, 32U);
            if (j + threadIdx.x < idx_next)
            {
                _shared_[threadIdx.x] = store_map_src[j + threadIdx.x];
            }
            mergeBitvectors(store, invCopy, pts_target * 32, numDstNodes, _shared_, toRel);
        }
    }
}

__global__ void kernel_insert_edges(const uint n, uint *from, uint *to, uint *ofst, uint *memory, int rel)
{
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    int stride = blockDim.y * gridDim.x;
    for (int i = index; i < n; i += stride)
    {

        uint offset = ofst[i];
        uint offset_next = ofst[i + 1];
        uint src = from[offset];

        for (size_t j = offset; j < offset_next; j++)
        {
            uint dst = to[j];
            insertEdgeDevice(src, dst, memory, rel);
        }
    }
}

__host__ void insertEdges(edgeSet *edges, uint *memory, int inv, int rel)
{
    uint *from, *to, *ofst, N;

    N = edges->second.size();

    checkCuda(hipMallocManaged(&from, N * sizeof(unsigned int)));
    checkCuda(hipMallocManaged(&to, N * sizeof(unsigned int)));
    checkCuda(hipMallocManaged(&ofst, N * sizeof(unsigned int)));

    if (inv)
    {
        memcpy(from, edges->second.data(), N * sizeof(unsigned int));
        memcpy(to, edges->first.data(), N * sizeof(unsigned int));
    }
    else
    {
        memcpy(from, edges->first.data(), N * sizeof(unsigned int));
        memcpy(to, edges->second.data(), N * sizeof(unsigned int));
    }

    thrust::sort_by_key(thrust::device, from, from + N, to);
    long numUnique = thrust::unique_by_key_copy(thrust::device, from, from + N, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), ofst).second - ofst;

    dim3 numBlocks(N_BLOCKS);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);

    checkCuda(hipDeviceSynchronize());
    kernel_insert_edges<<<numBlocks, threadsPerBlock>>>(numUnique, from, to, ofst, memory, rel);
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipFree(from));
    checkCuda(hipFree(to));
    checkCuda(hipFree(ofst));
}

/**
 * collect pts targets for src
 */
__host__ void collectFromBitvector(uint src, uint *memory, std::vector<uint> &pts)
{
    uint index = src * 32;
    uint base, next, bits, ptsTarget;

    while (index != UINT_MAX)
    {
        base = memory[index + BASE];
        next = memory[index + NEXT];

        if (base == UINT_MAX)
        {
            break;
        }

        for (size_t j = 0; j < BASE; j++)
        {
            bits = memory[index + j];
            for (size_t k = 0; k < 32; k++)
            {
                if (1 << k & bits)
                {
                    // calculate target from bitvector
                    ptsTarget = 960 * base + 32 * j + k;
                    pts.push_back(ptsTarget);
                }
            }
        }
        index = next;
    }
}

__host__ uint handleGepEdges(edgeSetOffset *gepEdges, uint *memory, void *consG, void *pag)
{
    for (size_t i = 0; i < gepEdges->second.size(); i++)
    {
        uint src, dst, offset, gepElement;
        src = gepEdges->first.first[i];
        offset = gepEdges->first.second[i];
        dst = gepEdges->second[i];

        std::vector<uint> targets;
        collectFromBitvector(src, memory, targets);
        for (uint target : targets)
        {
            gepElement = handleGep(consG, pag, target, offset);
            insertEdge(gepElement, dst, memory);
        }
    }
    uint nodeCount = getNodeCount(consG);
    return nodeCount;
}

__host__ bool alias(uint a, uint b, uint *memory)
{
    std::vector<uint> ptsA, ptsB;

    collectFromBitvector(a, memory, ptsA);
    collectFromBitvector(b, memory, ptsB);

    for (uint target : ptsA)
        if (std::find(ptsB.begin(), ptsB.end(), target) != ptsB.end())
            return true;

    return false;
}

__device__ void cloneAndLink(const uint var, const uint ptsIndex, uint &currDiffPtsIndex, const uint diffPtsBits, const uint diffPtsNext, uint *pts, uint *curr_pts, uint *next_pts)
{
    // clone(ptsIndex, diffPtsBits, diffPtsNext, PTS);
    insertBitvector(next_pts, pts, ptsIndex, diffPtsBits, PTS);
    if (currDiffPtsIndex != UINT_MAX)
    {
        curr_pts[currDiffPtsIndex + NEXT] = ptsIndex;
    }
    else
    {
        currDiffPtsIndex = 32 * var;
        uint ptsBits = pts[ptsIndex + threadIdx.x];
        curr_pts[currDiffPtsIndex + threadIdx.x] = ptsBits;
    }
}

/**
 * Update the current, next and total PTS sets of a variable. In the last iteration of the main
 * loop, points-to edges have been added to NEXT_DIFF_PTS. However, many of them might already be
 * present in PTS. The purpose of this function is to update PTS as PTS U NEXT_DIFF_PTS, and set
 * CURR_DIFF_PTS as the difference between the old and new PTS for the given variable.
 *
 * @param var ID of the variable
 * @param pts memory for all points to bitvectors
 * @param curr_pts memory for previous iterations points to bitvectors in working set
 * @param next_pts memory for all newly added pts bitvectors
 * @return true if new pts edges have been added to this variable
 */
__device__ bool updatePtsAndDiffPts(const uint var, uint *pts, uint *curr_pts, uint *next_pts)
{
    // next next index
    const uint diffPtsHeadIndex = var * 32;

    uint diffPtsBits = next_pts[diffPtsHeadIndex + threadIdx.x];
    uint diffPtsBase = __shfl_sync(0xFFFFFFFF, diffPtsBits, 30);

    if (diffPtsBase == UINT_MAX)
    {
        return false;
    }

    uint diffPtsNext = __shfl_sync(0xFFFFFFFF, diffPtsBits, 31);
    next_pts[diffPtsHeadIndex + threadIdx.x] = UINT_MAX;

    uint ptsIndex = var * 32;
    uint ptsBits = pts[ptsIndex + threadIdx.x];
    uint ptsBase = __shfl_sync(0xFFFFFFFF, ptsBits, 30);

    if (ptsBase == UINT_MAX)
    {
        // we pass ptsBase instead of UINT_MAX because it's also UINT_MAX but it can be modified
        cloneAndLink(var, ptsIndex, ptsBase, diffPtsBits, diffPtsNext, pts, curr_pts, next_pts);
        return true;
    }
    uint ptsNext = __shfl_sync(0xFFFFFFFF, ptsBits, 31);
    uint currDiffPtsIndex = UINT_MAX;
    while (1)
    {
        if (ptsBase > diffPtsBase)
        {
            uint newIndex = incEdgeCouter(PTS);
            pts[newIndex + threadIdx.x] = ptsBits;
            uint val = threadIdx.x == NEXT ? newIndex : diffPtsBits;
            pts[ptsIndex + threadIdx.x] = val;

            ptsIndex = newIndex;
            // update CURR_DIFF_PTS
            newIndex = currDiffPtsIndex == UINT_MAX ? 32 * var : incEdgeCouter(PTS_CURR);
            val = threadIdx.x == NEXT ? UINT_MAX : diffPtsBits;
            curr_pts[newIndex + threadIdx.x] = val;
            if (currDiffPtsIndex != UINT_MAX)
            {
                curr_pts[currDiffPtsIndex + NEXT] = newIndex;
            }
            if (diffPtsNext == UINT_MAX)
            {
                return true;
            }
            currDiffPtsIndex = newIndex;

            diffPtsBits = next_pts[diffPtsNext + threadIdx.x];
            diffPtsBase = __shfl_sync(0xFFFFFFFF, diffPtsBits, 30);
            diffPtsNext = __shfl_sync(0xFFFFFFFF, diffPtsBits, 31);
        }
        else if (ptsBase == diffPtsBase)
        {
            uint newPtsNext = (ptsNext == UINT_MAX && diffPtsNext != UINT_MAX) ? incEdgeCouter(PTS) : ptsNext;
            uint orBits = threadIdx.x == NEXT ? newPtsNext : ptsBits | diffPtsBits;
            uint ballot = __ballot_sync(0x3FFFFFFF, orBits != ptsBits);
            pts[ptsIndex + threadIdx.x] = orBits;
            if (ballot)
            {
                // update CURR_DIFF_PTS
                orBits = diffPtsBits & ~ptsBits;
                if (threadIdx.x == BASE)
                {
                    orBits = ptsBase;
                }
                else if (threadIdx.x == NEXT)
                {
                    orBits = UINT_MAX;
                }
                uint newIndex;
                if (currDiffPtsIndex != UINT_MAX)
                {

                    newIndex = incEdgeCouter(PTS_CURR);
                    curr_pts[currDiffPtsIndex + NEXT] = newIndex;
                }
                else
                {
                    newIndex = var * 32;
                }
                curr_pts[newIndex + threadIdx.x] = orBits;
                currDiffPtsIndex = newIndex;
            }
            if (diffPtsNext == UINT_MAX)
            {
                return (currDiffPtsIndex != UINT_MAX);
            }
            diffPtsBits = next_pts[diffPtsNext + threadIdx.x];
            diffPtsBase = __shfl_sync(0xFFFFFFFF, diffPtsBits, 30);
            diffPtsNext = __shfl_sync(0xFFFFFFFF, diffPtsBits, 31);

            if (ptsNext == UINT_MAX)
            {
                cloneAndLink(var, newPtsNext, currDiffPtsIndex, diffPtsBits, diffPtsNext, pts, curr_pts, next_pts);
                return true;
            }
            ptsIndex = ptsNext;

            ptsBits = pts[ptsIndex + threadIdx.x];
            ptsBase = __shfl_sync(0xFFFFFFFF, ptsBits, 30);
            ptsNext = __shfl_sync(0xFFFFFFFF, ptsBits, 31);
        }
        else
        { // ptsBase > diffPtsBase
            if (ptsNext == UINT_MAX)
            {
                uint newPtsIndex = incEdgeCouter(PTS);
                pts[ptsIndex + NEXT] = newPtsIndex;
                cloneAndLink(var, newPtsIndex, currDiffPtsIndex, diffPtsBits, diffPtsNext, pts, curr_pts, next_pts);
                return true;
            }
            ptsIndex = ptsNext;
            ptsBits = pts[ptsIndex + threadIdx.x];
            ptsBase = __shfl_sync(0xFFFFFFFF, ptsBits, 30);
            ptsNext = __shfl_sync(0xFFFFFFFF, ptsBits, 31);
        }
    }
}

__global__ void kernel_updatePts(const uint n, uint *pts, uint *curr_pts, uint *next_pts)
{
    if (__pts__ != pts)
    {
        __pts__ = pts;
        __ptsCurr__ = curr_pts;
        __ptsNext__ = next_pts;
    }

    __done__ = true;
    bool newWork = false;
    for (uint i = blockIdx.x * blockDim.y + threadIdx.y; i < n - 1; i += blockDim.y * gridDim.x)
    {
        bool newStuff = updatePtsAndDiffPts(i, pts, curr_pts, next_pts);
        newWork |= newStuff;
        if (!newStuff)
        {
            const uint currPtsHeadIndex = 32 * i;
            curr_pts[currPtsHeadIndex + threadIdx.x] = UINT_MAX;
        }
    }
    if (newWork)
    {
        __done__ = false;
    }
    __syncthreads();
    __freeList__[PTS_CURR] = n * 32;
    __freeList__[PTS_NEXT] = n * 32;
}

__host__ int run(unsigned int numNodes, edgeSet *addrEdges, edgeSet *directEdges, edgeSet *loadEdges, edgeSet *storeEdges, edgeSetOffset *gepEdges, void *consG, void *pag)
{
    int N = 1 << 28;
    uint *pts, *currPtsDiff, *nextPtsDiff, *invCopy, *invStore, *invLoad, *store_map_pts, *store_map_src, *store_map_idx;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&currPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&nextPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invCopy, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invStore, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invLoad, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&store_map_pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&store_map_src, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&store_map_idx, N * sizeof(uint1)));

    // set all values to UINT_MAX
    hipMemset(pts, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(currPtsDiff, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(nextPtsDiff, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(invCopy, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(invStore, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(invLoad, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_pts, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_src, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_idx, UCHAR_MAX, N * sizeof(unsigned int));

    // num of vertices
    size_t V{numNodes};

    // reserve 20% for new edges added by gep offsets
    uint initNum = std::ceil(1.2 * V) * ELEMENT_WIDTH;
    uint freeList[N_TYPES] = {initNum, initNum, initNum, initNum, initNum, initNum, initNum};
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(__freeList__), freeList, N_TYPES * sizeof(uint), 0, hipMemcpyHostToDevice));

    insertEdges(addrEdges, nextPtsDiff, 1, PTS_NEXT);
    insertEdges(directEdges, invCopy, 1, COPY);
    insertEdges(loadEdges, invLoad, 1, LOAD);
    insertEdges(storeEdges, invStore, 1, STORE);

    while (0)
    {
        dim3 numBlocks(N_BLOCKS);
        dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);

        kernel_updatePts<<<numBlocks, threadsPerBlock>>>(V, pts, currPtsDiff, nextPtsDiff);
        checkCuda(hipDeviceSynchronize());

        bool done = true;
        checkCuda(hipMemcpyFromSymbol(&done, HIP_SYMBOL(__done__), sizeof(bool)));

        if (done)
        {
            break;
        }
        kernel<<<numBlocks, threadsPerBlock>>>(V, invCopy, currPtsDiff, nextPtsDiff, PTS_NEXT);
        checkCuda(hipDeviceSynchronize());
        kernel<<<numBlocks, threadsPerBlock>>>(V, invLoad, currPtsDiff, invCopy, COPY);

        checkCuda(hipDeviceSynchronize());
        kernel_store<<<numBlocks, threadsPerBlock>>>(V, currPtsDiff, store_map_pts, store_map_src);
        checkCuda(hipDeviceSynchronize());

        thrust::sort_by_key(thrust::device, store_map_pts, store_map_pts + N, store_map_src);
        auto numSrcs = thrust::unique_by_key_copy(thrust::device, store_map_pts, store_map_pts + N, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), store_map_idx).second - store_map_idx;


        checkCuda(hipDeviceSynchronize());
        kernel_store2copy<<<numBlocks, threadsPerBlock>>>(numSrcs, store_map_pts, store_map_src, store_map_idx, pts, invStore, invCopy, COPY);
        checkCuda(hipDeviceSynchronize());
    }
    // Free memory
    checkCuda(hipFree(pts));
    checkCuda(hipFree(currPtsDiff));
    checkCuda(hipFree(nextPtsDiff));
    checkCuda(hipFree(invCopy));
    checkCuda(hipFree(invStore));
    checkCuda(hipFree(invLoad));
    checkCuda(hipFree(store_map_pts));
    checkCuda(hipFree(store_map_src));
    checkCuda(hipFree(store_map_idx));

    return 0;
}