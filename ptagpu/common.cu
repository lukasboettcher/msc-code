#include "hip/hip_runtime.h"
#include "common.cuh"
#include <iostream>
#include <bitset>

/**
 * getHeadIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 *
 * \return index of the
 *
 */
__host__ __device__ size_t getHeadIndex(uint src, uint *graph)
{
    return 0;
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ __device__ void insertEdge(uint src, uint dst, uint *graph)
{
    uint index = src * 32;
    if (graph[index + BASE] == UINT_MAX)
        for (size_t i = 0; i < ELEMENT_WIDTH - 1; i++)
            graph[index + i] = 0;

    graph[index] |= 1 << dst;
}

__host__ __device__ uint incEdgeCouter()
{
    __shared__ volatile uint _shared_[THREADS_PER_BLOCK / WARP_SIZE];
    if (threadIdx.x == 0)
    {
        _shared_[threadIdx.y] = atomicAdd(&__ptsFreeList__, 32);
    }
    return _shared_[threadIdx.y];
}

__device__ uint __ptsFreeList__;

__global__ void kernel(int n, uint *A, uint *B, uint *C)
{
    // each warp gets a shared block for one access to global memory
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 128];
    uint *const _shared_ = &_sh_[threadIdx.y * 128];
    for (uint src = blockIdx.x * blockDim.x + threadIdx.y; src < n; src += blockDim.x * gridDim.x)
    {
        uint index = src * 32;
        do
        {
            uint bits = A[index + threadIdx.x];
            uint base = A[index + BASE];
            if (base == UINT_MAX)
                break;
            // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
            uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
            const uint threadMask = 1 << threadIdx.x;
            const uint myMask = threadMask - 1;
            while (nonEmptyThreads)
            {
                // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
                int leastThread = __ffs(nonEmptyThreads) - 1;
                // remove lsb from nonEmptyThreads (iteration step)
                nonEmptyThreads &= (nonEmptyThreads - 1);
                // share current bits with all threads in warp
                uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);

                // use the base and the word of the current thread's bits to calculate the target dst id
                uint var = base * 30 * 32 + 32 * leastThread + threadIdx.x;
                // check if this thread is looking at a dst node
                // uint bitActive = (var != 1U) && (current_bits & threadMask);
                uint bitActive = (current_bits & threadMask);
                // count threads that are looking at dst nodes
                uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
                uint numDstNodes = __popc(threadsWithDstNode);
                // calculate pos in shared mem, by counting prev threads that had a dst node
                uint pos = 0 + __popc(threadsWithDstNode & myMask);
                if (bitActive)
                {
                    _shared_[pos] = var;
                }
                if (numDstNodes)
                {
                    // go through all dst nodes, and union the out edges of that node w/ src's out nodes
                    for (size_t i = 0; i < numDstNodes; i++)
                    {
                        uint fromDstNode = _shared_[i];
                        uint fromIndex = fromDstNode * 32;
                        // read dst out edges
                        uint fromBits = B[fromIndex + threadIdx.x];
                        uint fromBase = B[fromIndex + BASE];
                        if (fromBase == UINT_MAX)
                            continue;

                        uint fromNext = B[fromIndex + NEXT];
                        uint toIndex = index;
                        uint toBits = C[toIndex + threadIdx.x];
                        uint toBase = C[toIndex + BASE];
                        uint toNext = C[toIndex + NEXT];

                        if (toBase == UINT_MAX)
                            C[toIndex + threadIdx.x] = fromBits;
                        while (1)
                        {
                            if (toBase == fromBase)
                            {
                                // if target next is undefined, create new edge for more edges
                                uint newToNext = (toNext == UINT_MAX && fromNext != UINT_MAX) ? incEdgeCouter() : toNext;
                                // union the bits, adding the new edge
                                uint orBits = fromBits | toBits;
                                uint newBits = threadIdx.x == NEXT ? newToNext : orBits;
                                if (newBits != toBits)
                                {
                                    C[toIndex + threadIdx.x] = newBits;
                                }
                                if (fromNext == UINT_MAX)
                                {
                                    break;
                                }
                                fromBits = C[fromNext + threadIdx.x];
                                fromBase = C[fromNext + BASE];
                                fromNext = C[fromNext + NEXT];
                                if (toNext == UINT_MAX)
                                {
                                    while (1)
                                    {
                                        uint newIndex = fromNext == UINT_MAX ? UINT_MAX : incEdgeCouter();
                                        uint val = threadIdx.x == NEXT ? newIndex : fromBits;
                                        C[toIndex + threadIdx.x] = val;
                                        if (fromNext == UINT_MAX)
                                        {
                                            break;
                                        }
                                        toIndex = newIndex;
                                        fromBits = C[fromNext + threadIdx.x];
                                        fromNext = C[fromNext + NEXT];
                                    }
                                    break;
                                }
                                toIndex = newToNext;
                                toBits = C[toNext + threadIdx.x];
                                toBase = C[toNext + BASE];
                                toNext = C[toNext + NEXT];
                            }
                        }
                    }
                }
            }

            index = A[index + NEXT];
        } while (index != UINT_MAX);
    }
}

__host__ int run()
{
    // CUDA kernel to add elements of two arrays

    int N = 1 << 20;
    uint *pts, *prevPtsDiff, *currPtsDiff, *invCopy, *invStore, *invLoad;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&prevPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&currPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invCopy, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invStore, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invLoad, N * sizeof(uint1)));

    // set all values to UINT_MAX
    hipMemset(pts, UINT_MAX, N);
    hipMemset(prevPtsDiff, UINT_MAX, N);
    hipMemset(currPtsDiff, UINT_MAX, N);
    hipMemset(invCopy, UINT_MAX, N);
    hipMemset(invStore, UINT_MAX, N);
    hipMemset(invLoad, UINT_MAX, N);

    // insertEdge(1, 0, pts);
    // insertEdge(2, 1, invLoad);
    // insertEdge(1, 3, invStore);
    // insertEdge(3, 4, pts);
    insertEdge(0, 1, invCopy);
    insertEdge(1, 2, pts);

    // num of vertices
    size_t V{3};

    uint numPtsElementsFree = V * ELEMENT_WIDTH;
    hipMemcpyToSymbol(HIP_SYMBOL(__ptsFreeList__), &numPtsElementsFree, sizeof(uint));

    dim3 numBlocks(16);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);
    kernel<<<numBlocks, threadsPerBlock>>>(V, invCopy, pts, pts);

    // Wait for GPU to finish before accessing on host
    checkCuda(hipDeviceSynchronize());

    // Free memory
    checkCuda(hipFree(pts));
    checkCuda(hipFree(prevPtsDiff));
    checkCuda(hipFree(currPtsDiff));
    checkCuda(hipFree(invCopy));
    checkCuda(hipFree(invStore));
    checkCuda(hipFree(invLoad));

    return 0;
}