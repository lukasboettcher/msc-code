#include "hip/hip_runtime.h"
#include "common.cuh"

std::map<unsigned int *, unsigned int> testMap;

/**
 * __ptsFreeList__
 * this is the head of the free list
 * keeps track of last allocated memory location
 * access needs to be atomic to prevent collisions
 *
 */
__device__ uint __freeList__[N_TYPES];

/**
 * getHeadIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 *
 * \return index of the
 *
 */
__host__ __device__ size_t getHeadIndex(uint src, uint *graph)
{
    return 0;
}

__device__ uint incEdgeCouter(int type)
{
    __shared__ volatile uint _shared_[THREADS_PER_BLOCK / WARP_SIZE];
    if (threadIdx.x == 0)
    {
        _shared_[threadIdx.y] = atomicAdd(&__freeList__[type], 32);
    }
    return _shared_[threadIdx.y];
}

__device__ uint insertEdgeDevice(uint src, uint dst, uint *graph, uint toRel)
{
    uint index = src * 32;
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    uint myBits = 0;

    if (threadIdx.x == word)
        myBits = 1 << bit;
    else if (threadIdx.x == BASE)
        myBits = base;
    else if (threadIdx.x == NEXT)
        myBits = UINT_MAX;

    while (1)
    {
        uint toBits = graph[index + threadIdx.x];
        uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
        if (toBase == UINT_MAX)
        {
            graph[index + threadIdx.x] = myBits;
            return index;
        }
        if (toBase == base)
        {
            uint orBits = toBits | myBits;
            if (orBits != toBits && threadIdx.x < NEXT)
                graph[index + threadIdx.x] = orBits;

            return index;
        }
        if (toBase < base)
        {
            uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            if (toNext == UINT_MAX)
            {
                uint newIndex = incEdgeCouter(toRel);
                graph[index + NEXT] = newIndex;
                graph[newIndex + threadIdx.x] = myBits;
                return newIndex;
            }
            index = toNext;
        }
        else
        {
            uint newIndex = incEdgeCouter(toRel);
            graph[newIndex + threadIdx.x] = toBits;
            uint val = threadIdx.x == NEXT ? newIndex : myBits;
            graph[index + threadIdx.x] = val;
            return index;
        }
    }
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ void insertEdge(uint src, uint dst, uint *graph)
{
    uint index = src * 32;
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    // printf("inserting edge %u -> %u\n\tindex: %u\n\tbase: %u\n\tword: %u\n\tbit: %u\n", src, dst, index, base, word, bit);

    if (graph[index + BASE] == UINT_MAX)
    {
        for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
            graph[index + i] = 0;
        graph[index + BASE] = base;
        graph[index + word] |= 1 << bit;
        return;
    }

    while (1)
    {
        uint toBase = graph[index + BASE];
        uint toNext = graph[index + NEXT];

        if (toBase == UINT_MAX)
        {
            for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                graph[index + i] = 0;
            graph[index + BASE] = base;
            graph[index + word] |= 1 << bit;
            return;
        }
        if (toBase < base)
        {
            if (toNext == UINT_MAX)
            {
                uint nextIndex = ++testMap[graph] * 32;
                graph[index + NEXT] = nextIndex;

                for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                    graph[nextIndex + i] = 0;
                graph[nextIndex + BASE] = base;
                graph[nextIndex + word] |= 1 << bit;

                return;
            }

            index = toNext;
        }
        else if (base == toBase)
        {
            graph[index + word] |= 1 << bit;
            return;
        }
        else if (toBase > base)
        {

            uint nextIndex = ++testMap[graph] * 32;
            for (size_t i = 0; i < ELEMENT_WIDTH; i++)
                graph[nextIndex + i] = graph[index + i];
            for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
                graph[nextIndex + i] = 0;
            graph[index + BASE] = base;
            graph[index + NEXT] = nextIndex;
            graph[index + word] |= 1 << bit;
        }
    }
}

__device__ void insertBitvector(uint *originMemory, uint *targetMemory, uint toIndex, uint fromBits, uint toRel)
{
    while (1)
    {
        // use warp intrinsics to get next index in from memory
        uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
        // check if a new bitvector is required
        // if that is the case, allocate a new index for a new element
        uint toNext = fromNext == UINT_MAX ? UINT_MAX : incEdgeCouter(toRel);
        // handle the special next entry, since we can not reuse the fromNext bits
        uint val = threadIdx.x == NEXT ? toNext : fromBits;
        // write new values to target memory location
        targetMemory[toIndex + threadIdx.x] = val;
        // exit if no more elements in from bitvector
        if (fromNext == UINT_MAX)
            return;
        toIndex = toNext;
        fromBits = originMemory[fromNext + threadIdx.x];
    }
}

__device__ void mergeBitvectors(uint *A, uint *B, uint *C, uint index, uint numDstNodes, uint *const _shared_, uint toRel)
{
    // go through all dst nodes, and union the out edges of that node w/ src's out nodes
    for (size_t i = 0; i < numDstNodes; i++)
    {
        uint fromIndex = _shared_[i] * 32;
        // read dst out edges
        uint fromBits = B[fromIndex + threadIdx.x];
        // get the base from thread nr 30
        uint fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
        // terminate if no data in from from bitvector
        if (fromBase == UINT_MAX)
            continue;
        // get the next index from thread nr 31
        uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);

        // share needed data for to indices
        uint toIndex = index;
        uint toBits = C[toIndex + threadIdx.x];
        uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
        uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);

        if (toBase == UINT_MAX)
        {
            insertBitvector(B, C, toIndex, fromBits, toRel);
            continue;
        }

        while (1)
        {
            if (toBase == fromBase)
            {
                // if target next is undefined, create new edge for more edges
                uint newToNext = (toNext == UINT_MAX && fromNext != UINT_MAX) ? incEdgeCouter(toRel) : toNext;
                // union the bits, adding the new edges
                uint orBits = fromBits | toBits;
                // each thread gets a value that will be written back to memory
                uint val = threadIdx.x == NEXT ? newToNext : orBits;
                if (val != toBits)
                {
                    C[toIndex + threadIdx.x] = val;
                }

                // if no more bitvectors in origin, end loop
                if (fromNext == UINT_MAX)
                {
                    break;
                }
                // else load next bits
                fromBits = C[fromNext + threadIdx.x];
                fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
                fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
                if (toNext == UINT_MAX)
                {
                    insertBitvector(B, C, toIndex, fromBits, toRel);
                    break;
                }
                toIndex = newToNext;
                toBits = C[toNext + threadIdx.x];
                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            }
            else if (toBase < fromBase)
            {
                // if toNext is undefined, we need to allocate a new element
                // after that, we can simply insert teh origin bitvector
                if (toNext == UINT_MAX)
                {
                    toNext = incEdgeCouter(toRel);
                    insertBitvector(B, C, toNext, fromBits, toRel);
                    break;
                }
                // if toNext is defined, load those to bits for the next iteration
                toIndex = toNext;
                toBits = C[toNext + threadIdx.x];
                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            }
            else if (toBase > fromBase)
            {
                // if toBase is greater than frombase
                // we need to insert enother bitvector element before toindex
                // and shift the current element back (ref. linked lists)
                uint newIndex = incEdgeCouter(toRel);
                // write the current bits from the target element to a new location
                C[newIndex + threadIdx.x] = toBits;
                // then overwrite the current bits with fromBits (insert before node)
                uint val = threadIdx.x == NEXT ? newIndex : fromBits;
                C[toIndex + threadIdx.x] = val;

                // if next from element is defined, update the bits
                // if not, break for this element
                if (fromNext == UINT_MAX)
                    break;

                toIndex = newIndex;

                fromBits = C[fromNext + threadIdx.x];
                fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
                fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
            }
        }
    }
}

__global__ void kernel(int n, uint *A, uint *B, uint *C, uint toRel)
{
    // each warp gets a shared block for one access to global memory
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 128];
    uint *const _shared_ = &_sh_[threadIdx.y * 128];
    for (uint src = blockIdx.x * blockDim.x + threadIdx.y; src < n; src += blockDim.x * gridDim.x)
    {
        uint index = src * 32;
        do
        {
            uint bits = A[index + threadIdx.x];
            uint base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;
            // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
            uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
            const uint threadMask = 1 << threadIdx.x;
            const uint myMask = threadMask - 1;
            while (nonEmptyThreads)
            {
                // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
                int leastThread = __ffs(nonEmptyThreads) - 1;
                // remove lsb from nonEmptyThreads (iteration step)
                nonEmptyThreads &= (nonEmptyThreads - 1);
                // share current bits with all threads in warp
                uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);

                // use the base and the word of the current thread's bits to calculate the target dst id
                uint var = base * 30 * 32 + 32 * leastThread + threadIdx.x;
                // check if this thread is looking at a dst node
                // uint bitActive = (var != 1U) && (current_bits & threadMask);
                uint bitActive = (current_bits & threadMask);
                // count threads that are looking at dst nodes
                uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
                uint numDstNodes = __popc(threadsWithDstNode);
                // calculate pos in shared mem, by counting prev threads that had a dst node
                uint pos = 0 + __popc(threadsWithDstNode & myMask);
                if (bitActive)
                {
                    _shared_[pos] = var;
                }
                if (numDstNodes)
                {
                    mergeBitvectors(A, B, C, index, numDstNodes, _shared_, toRel);
                }
            }
            index = __shfl_sync(0xFFFFFFFF, bits, 31);
        } while (index != UINT_MAX);
    }
}

__device__ uint store_map_head = 0;

__device__ void insert_store_map(const uint src, const uint n, uint *const list, uint *store_map_pts, uint *store_map_src)
{
    for (int i = 0; i < n; i += 32)
    {
        uint size = min(n - i, 32);
        uint next;
        if (!threadIdx.x)
        {
            next = atomicAdd(&store_map_head, size);
        }
        next = __shfl_sync(0xFFFFFFFF, next, 0);
        if (threadIdx.x < size)
        {
            store_map_pts[next + threadIdx.x] = list[i + threadIdx.x];
            // store_map_src[next + threadIdx.x] = src_index;
            store_map_src[next + threadIdx.x] = src;
        }
    }
}

/**
 * Kernel for store edges,
 * here we need to collect all store edges that share a pts edge
 * and selectively assign them to the same warps
 * so that we save on synchronization between warps
 */
__global__ void kernel_store(int n, uint *A, uint *B, uint *C)
{
    // each warp gets a shared block for one access to global memory
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 128];
    uint *const _shared_ = &_sh_[threadIdx.y * 128];
    for (uint src = blockIdx.x * blockDim.x + threadIdx.y; src < n; src += blockDim.x * gridDim.x)
    {
        uint index = src * 32;
        uint usedShared = 0;
        do
        {
            uint bits = A[index + threadIdx.x];
            uint base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;
            // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
            uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
            const uint threadMask = 1 << threadIdx.x;
            const uint myMask = threadMask - 1;
            while (nonEmptyThreads)
            {
                // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
                int leastThread = __ffs(nonEmptyThreads) - 1;
                // remove lsb from nonEmptyThreads (iteration step)
                nonEmptyThreads &= (nonEmptyThreads - 1);
                // share current bits with all threads in warp
                uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);

                // use the base and the word of the current thread's bits to calculate the target dst id
                uint var = base * 30 * 32 + 32 * leastThread + threadIdx.x;
                // check if this thread is looking at a dst node
                // uint bitActive = (var != 1U) && (current_bits & threadMask);
                uint bitActive = (current_bits & threadMask);
                // count threads that are looking at dst nodes
                uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
                uint numDstNodes = __popc(threadsWithDstNode);
                if (usedShared + numDstNodes > 128)
                {
                    insert_store_map(index, usedShared, _shared_, B, C);
                    usedShared = 0;
                }
                // calculate pos in shared mem, by counting prev threads that had a dst node
                uint pos = usedShared + __popc(threadsWithDstNode & myMask);
                if (bitActive)
                {
                    _shared_[pos] = var;
                }
                usedShared += numDstNodes;
            }
            index = __shfl_sync(0xFFFFFFFF, bits, 31);
        } while (index != UINT_MAX);
        if (usedShared)
        {
            insert_store_map(src, usedShared, _shared_, B, C);
        }
    }
}

__global__ void kernel_store2copy(const uint n, uint *store_map_pts, uint *store_map_src, uint *store_map_idx, uint *pts, uint *store, uint *invCopy, uint toRel)
{
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 128];
    uint *const _shared_ = &_sh_[threadIdx.y * 128];
    for (uint i = blockIdx.x * blockDim.x + threadIdx.y; i < n - 1; i += blockDim.x * gridDim.x)
    {
        uint idx = store_map_idx[i];
        uint idx_next = store_map_idx[i + 1];

        // load the pts target, this should no change for the next totalDstNodes
        uint pts_target = store_map_pts[idx];

        for (uint j = idx; j < idx_next; j += 32)
        {
            uint numDstNodes = min(idx_next - j, 32U);
            if (j + threadIdx.x < idx_next)
            {
                _shared_[threadIdx.x] = store_map_src[j + threadIdx.x];
            }
            mergeBitvectors(pts, store, invCopy, pts_target * 32, numDstNodes, _shared_, toRel);
        }
    }
}

__global__ void kernel_insert_edges(const uint n, uint *from, uint *to, uint *ofst, uint *memory, int rel)
{
    int index = blockIdx.x * blockDim.x + threadIdx.y;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {

        uint offset = ofst[i];
        uint offset_next = ofst[i + 1];
        uint src = from[offset];

        for (size_t j = offset; j < offset_next; j++)
        {
            uint dst = to[j];
            insertEdgeDevice(src, dst, memory, rel);
        }
    }
}

__host__ void insertEdges(edgeSet *edges, uint *memory, int inv, int rel)
{
    uint *from, *to, *ofst, N;

    N = edges->second.size();

    checkCuda(hipMallocManaged(&from, N * sizeof(unsigned int)));
    checkCuda(hipMallocManaged(&to, N * sizeof(unsigned int)));
    checkCuda(hipMallocManaged(&ofst, N * sizeof(unsigned int)));

    if (inv)
    {
        memcpy(from, edges->second.data(), N * sizeof(unsigned int));
        memcpy(to, edges->first.data(), N * sizeof(unsigned int));
    }
    else
    {
        memcpy(from, edges->first.data(), N * sizeof(unsigned int));
        memcpy(to, edges->second.data(), N * sizeof(unsigned int));
    }

    thrust::sort_by_key(thrust::device, from, from + N, to);
    long numUnique = thrust::unique_by_key_copy(thrust::device, from, from + N, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), ofst).second - ofst;

    // CUDA kernel to add elements of two arrays

    dim3 numBlocks(16);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);

    checkCuda(hipDeviceSynchronize());
    kernel_insert_edges<<<numBlocks, threadsPerBlock>>>(numUnique, from, to, ofst, memory, rel);
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipFree(from));
    checkCuda(hipFree(to));
    checkCuda(hipFree(ofst));
}

__host__ int run(unsigned int numNodes, edgeSet *addrEdges, edgeSet *directEdges, edgeSet *loadEdges, edgeSet *storeEdges, edgeSetOffset *gepEdges)
{
    int N = 1 << 28;
    uint *pts, *prevPtsDiff, *currPtsDiff, *invCopy, *invStore, *invLoad, *store_map_pts, *store_map_src, *store_map_idx;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&prevPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&currPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invCopy, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invStore, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invLoad, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&store_map_pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&store_map_src, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&store_map_idx, N * sizeof(uint1)));

    // set all values to UINT_MAX
    hipMemset(pts, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(prevPtsDiff, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(currPtsDiff, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(invCopy, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(invStore, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(invLoad, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_pts, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_src, UCHAR_MAX, N * sizeof(unsigned int));
    hipMemset(store_map_idx, UCHAR_MAX, N * sizeof(unsigned int));

    // num of vertices
    size_t V{numNodes};

    // reserve 20% for new edges added by gep offsets
    uint initNum = std::ceil(1.2 * V) * ELEMENT_WIDTH;
    uint freeList[N_TYPES] = {initNum, initNum, initNum, initNum, initNum};
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(__freeList__), freeList, N_TYPES * sizeof(uint), 0, hipMemcpyHostToDevice));

    insertEdges(addrEdges, pts, 1, PTS);
    insertEdges(directEdges, invCopy, 1, COPY);
    insertEdges(loadEdges, invLoad, 1, LOAD);
    insertEdges(storeEdges, invStore, 1, STORE);

    for (size_t i = 0; i < 10; i++)
    {
        dim3 numBlocks(16);
        dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);
        kernel<<<numBlocks, threadsPerBlock>>>(V, invCopy, pts, pts, PTS);
        checkCuda(hipDeviceSynchronize());
        kernel<<<numBlocks, threadsPerBlock>>>(V, invLoad, pts, invCopy, COPY);

        checkCuda(hipDeviceSynchronize());
        kernel_store<<<numBlocks, threadsPerBlock>>>(V, pts, store_map_pts, store_map_src);
        checkCuda(hipDeviceSynchronize());

        thrust::sort_by_key(thrust::device, store_map_pts, store_map_pts + N, store_map_src);
        auto numSrcs = thrust::unique_by_key_copy(thrust::device, store_map_pts, store_map_pts + N, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), store_map_idx).second - store_map_idx;


        checkCuda(hipDeviceSynchronize());
        kernel_store2copy<<<numBlocks, threadsPerBlock>>>(numSrcs, store_map_pts, store_map_src, store_map_idx, pts, invStore, invCopy, COPY);
        checkCuda(hipDeviceSynchronize());
    }
    // Free memory
    checkCuda(hipFree(pts));
    checkCuda(hipFree(prevPtsDiff));
    checkCuda(hipFree(currPtsDiff));
    checkCuda(hipFree(invCopy));
    checkCuda(hipFree(invStore));
    checkCuda(hipFree(invLoad));
    checkCuda(hipFree(store_map_pts));
    checkCuda(hipFree(store_map_src));
    checkCuda(hipFree(store_map_idx));

    return 0;
}