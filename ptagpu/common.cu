#include "hip/hip_runtime.h"
#include "common.cuh"

/**
 *
 * this variable holds strings for each of the relations
 * this allows us to get a name from a numeric relation
 *
 */
char const *relNames[6] = {"PTS", "PTS Current", "PTS Next", "Inv COPY", "Inv LOAD", "Inv Store"};

struct KernelInfo
{
    bool initialized = false;
    dim3 blockSize;
    dim3 gridSize;
    size_t sharedMemory;
    float elapsedTime = 0;
    hipEvent_t start, stop;
};

/**
 *
 * store calculated kernel parameters here
 *
 */
std::map<void *, KernelInfo> kernelParameters;

/**
 * represents the number of nodes in the graph
 *
 */
__device__ __managed__ size_t V;

/**
 * keeps track of last allocated memory location
 * for each memory region / relation
 * access needs to be atomic to prevent collisions
 *
 */
__device__ __managed__ index_t __freeList__[N_TYPES];

/**
 * is a temporary counter for statistics,
 * since freelist for currpts is reset after updatePts
 *
 */
__device__ __managed__ index_t tmpFreePtsCurr = 0;

/**
 *
 * this variable represents to max number of nodes
 *
 * is to be initialized with enough overhead
 * to allow adding further nodes via gep offsets calculations
 * currently set to 120% of constraint graph
 *
 */
__device__ __managed__ uint __reservedHeader__;

/**
 *
 * flag that keeps track of remaining work
 * if true, no next iteration needed
 *
 */
__device__ __managed__ bool __done__ = true;

/**
 *
 * these variables are used throughout the
 * code for hashmap operations
 * where keys are associated w/ values
 * and then sorted / uniqued
 *
 */
__device__ __managed__ uint *__key__;

/**
 *
 * these variables are used throughout the
 * code for hashmap operations
 * where keys are associated w/ values
 * and then sorted / uniqued
 *
 */
__device__ __managed__ uint *__val__;

/**
 *
 * these variables are used throughout the
 * code for hashmap operations
 * where keys are associated w/ values
 * and then sorted / uniqued
 *
 */
__device__ __managed__ uint *__offsets__;

/**
 * __numKeys__
 *
 * as kv pair memory is allocated once
 * with overhead, __numKeys__ keeps track
 * of actually used space to prevent
 * redundant work
 *
 */
__device__ __managed__ uint __numKeys__;

/**
 *
 * various counters for wortklist algorithms
 * adapted from mendez et. al
 *
 */
__device__ __managed__ uint __counter__ = 0;

/**
 *
 * various counters for wortklist algorithms
 * adapted from mendez et. al
 *
 */
__device__ uint __worklistIndex0__ = 0;

/**
 *
 * various counters for wortklist algorithms
 * adapted from mendez et. al
 *
 */
__device__ uint __worklistIndex1__ = 0;

/**
 *
 * is used as a freelist for the kv store
 *
 */
__device__ uint __storeMapHead__ = 0;

/**
 *
 * device pointers for the pts bitvectors
 * these need to be accesses adhoc
 * so are written to device symbols permanently
 *
 */
__device__ __managed__ uint *__memory__;

/**
 *
 * alternative memory location
 * for store constraints
 * separate from main memory
 * for store inv kernel
 *
 */
__device__ __managed__ uint *__storeConstraints__;

/**
 *
 * is the corresponding counter
 *
 */
__device__ __managed__ uint __numStoreConstraints__;

/**
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 * \param rel relation for which to get the head index
 *
 * \return index of the bitvector
 *
 */
__host__ __device__ index_t getIndex(uint src, uint rel)
{
    switch (rel)
    {
    case PTS:
        return OFFSET_PTS + (ELEMENT_WIDTH * src);
    case PTS_CURR:
        return OFFSET_PTS_CURR + (ELEMENT_WIDTH * src);
    case PTS_NEXT:
        return OFFSET_PTS_NEXT + (ELEMENT_WIDTH * src);
    case COPY:
        return OFFSET_COPY + (ELEMENT_WIDTH * src);
    case LOAD:
        return OFFSET_LOAD + (ELEMENT_WIDTH * src);
    case STORE:
        return OFFSET_STORE + (ELEMENT_WIDTH * src);
    }
    return src * ELEMENT_WIDTH;
}

/**
 *
 * increment the edge counter and return free memory location
 * for new element from the host
 *
 * \param type relation for which to reserve memory
 *
 * \return index for new bitvector elememt
 *
 */
__host__ index_t incEdgeCouterHost(int type)
{
    index_t index = __freeList__[type];
    __freeList__[type] += 32;
    return index;
}

/**
 *
 * increment the edge counter and return free memory location
 * for new element from the device
 *
 * \param type relation for which to reserve memory
 *
 * \return index for new bitvector elememt
 *
 */
__device__ inline index_t incEdgeCouter(int type)
{
    index_t newIndex;
    if (!threadIdx.x)
        newIndex = atomicAdd_system(&__freeList__[type], 32);
    newIndex = __shfl_sync(FULL_MASK, newIndex, 0);
    return newIndex;
}

/**
 *
 * insert an edge into the right memory location from device
 *
 * \param src edge src
 * \param dst edge dst
 * \param toRel edge type
 *
 * \return index for inserted edge
 *
 */
__device__ index_t insertEdgeDevice(uint src, uint dst, uint toRel)
{
    index_t index = getIndex(src, toRel);
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    uint myBits = 0;

    if (threadIdx.x == word)
        myBits = 1 << bit;
    else if (threadIdx.x == BASE)
        myBits = base;
    else if (threadIdx.x == NEXT_LOWER)
        myBits = UINT_MAX;
    else if (threadIdx.x == NEXT_UPPER)
        myBits = UINT_MAX;

    while (1)
    {
        uint toBits = __memory__[index + threadIdx.x];
        uint toBase = __shfl_sync(FULL_MASK, toBits, BASE);
        if (toBase == UINT_MAX)
        {
            __memory__[index + threadIdx.x] = myBits;
            return index;
        }
        else if (toBase == base)
        {
            uint orBits = toBits | myBits;
            if (orBits != toBits && threadIdx.x < NEXT_LOWER)
                __memory__[index + threadIdx.x] = orBits;

            return index;
        }
        else if (toBase < base)
        {
            index_t toNext = thread_load_size_t(toBits);
            if (toNext == ULLONG_MAX)
            {
                index_t newIndex = incEdgeCouter(toRel);
                store_size_t(__memory__, index, newIndex);
                __memory__[newIndex + threadIdx.x] = myBits;
                return newIndex;
            }
            index = toNext;
        }
        else if (toBase > base)
        {
            index_t newIndex = incEdgeCouter(toRel);
            __memory__[newIndex + threadIdx.x] = toBits;
            uint val = thread_load_val(myBits, newIndex);
            __memory__[index + threadIdx.x] = val;
            return index;
        }
    }
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ void insertEdge(uint src, uint dst, uint *graph, uint toRel)
{
    index_t index = getIndex(src, toRel);
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);

    if (graph[index + BASE] == UINT_MAX)
    {
        for (size_t i = 0; i < BASE; i++)
            graph[index + i] = 0;
        graph[index + BASE] = base;
        graph[index + word] |= 1 << bit;
        return;
    }

    while (1)
    {
        uint toBase = graph[index + BASE];
        index_t toNext = load_size_t(graph[index + NEXT_LOWER], graph[index + NEXT_UPPER]);

        if (toBase == UINT_MAX)
        {
            for (size_t i = 0; i < BASE; i++)
                graph[index + i] = 0;
            graph[index + BASE] = base;
            graph[index + word] |= 1 << bit;
            return;
        }
        if (toBase < base)
        {
            if (toNext == ULLONG_MAX)
            {
                index_t nextIndex = incEdgeCouterHost(toRel);
                store_size_t(graph, index, nextIndex);

                for (size_t i = 0; i < BASE; i++)
                    graph[nextIndex + i] = 0;
                graph[nextIndex + BASE] = base;
                graph[nextIndex + word] |= 1 << bit;
                return;
            }

            index = toNext;
        }
        else if (base == toBase)
        {
            graph[index + word] |= 1 << bit;
            return;
        }
        else if (toBase > base)
        {

            index_t nextIndex = incEdgeCouterHost(toRel);
            for (size_t i = 0; i < ELEMENT_WIDTH; i++)
                graph[nextIndex + i] = graph[index + i];
            for (size_t i = 0; i < BASE; i++)
                graph[nextIndex + i] = 0;
            graph[index + BASE] = base;
            store_size_t(graph, index, nextIndex);
            graph[index + word] |= 1 << bit;
        }
    }
}

template <uint fromRel, uint toRel>
__device__ void mergeBitvectors(const uint to, const uint numDstNodes, uint *_shared_);

template <uint fromRel, uint toRel>
__device__ void collectBitvectorTargets(const uint to, const uint bits, const uint base, uint *storage, uint &usedStorage);

/**
 *
 * helper function to increment counter
 * used in worklist algorithms
 * syncd via warp intrinsics
 *
 * \param counter counter to be incremented
 * \param delta amount to increment
 *
 * \return previous counter value
 *
 */
__device__ inline uint getAndIncrement(uint *counter, uint delta)
{
    uint cnt;
    if (!threadIdx.x)
        cnt = atomicAdd_system(counter, delta);
    cnt = __shfl_sync(FULL_MASK, cnt, 0);
    return cnt;
}

/**
 *
 * helper function to reset worklist counters
 * after operation
 * syncs blocks and grid by using counter
 *
 *
 * \return boolean whether thread is grid leader
 *
 */
__device__ inline uint resetWorklistIndex()
{
    __syncthreads();
    if (!threadIdx.x && !threadIdx.y && atomicInc_system(&__counter__, gridDim.x - 1) == (gridDim.x - 1))
    {
        __worklistIndex0__ = 0;
        __counter__ = 0;
        return 1;
    }
    return 0;
}

/**
 *
 * insert store, pointer pair into kv store
 *
 * \param src corresponding store src node
 * \param _shared_ passed shared memory containing nodes
 * \param numFrom numver of values in shared memory
 *
 * \return previous counter value
 *
 */
__device__ void insert_store_map(const uint src, uint *const _shared_, uint numFrom)
{
    const index_t storeIndex = getIndex(src, STORE);
    for (int i = 0; i < numFrom; i += 32)
    {
        uint size = min(numFrom - i, 32);
        uint next = getAndIncrement(&__storeMapHead__, size);
        if (threadIdx.x < size)
        {
            __key__[next + threadIdx.x] = _shared_[i + threadIdx.x];
            __val__[next + threadIdx.x] = src;
        }
    }
}

/**
 *
 * merges two bitvectors and optionally applies copy
 * pointer operations
 *
 * \param to target node
 * \param fromIndex index of second rel nodes, to be merged
 * \param storage shared memory shard for recursive mergeBV call
 * \param applyCopy should always be true, apply copy rule?
 *
 */
__device__ void mergeBitvectorCopy(const uint to, const index_t fromIndex, uint *const storage, bool applyCopy = true)
{
    index_t toIndex = getIndex(to, COPY);
    if (fromIndex == toIndex)
    {
        return;
    }
    // read dst out edges
    uint fromBits = __memory__[fromIndex + threadIdx.x];
    // get the base from thread
    uint fromBase = __shfl_sync(FULL_MASK, fromBits, BASE);
    // terminate if no data in from bitvector
    if (fromBase == UINT_MAX)
    {
        return;
    }
    // get the next index from thread
    index_t fromNext = thread_load_size_t(fromBits);

    // share needed data for to indices
    uint toBits = __memory__[toIndex + threadIdx.x];
    uint toBase = __shfl_sync(FULL_MASK, toBits, BASE);
    index_t toNext = thread_load_size_t(toBits);

    // keep count of used storage in shared memory
    // this storage is adjacent to previous collectBitvectorTargets memory
    uint numFrom = 0;
    index_t newVal;
    while (1)
    {

        if (toBase == fromBase)
        {
            // union the bits, adding the new edges
            uint orBits = fromBits | toBits;
            uint diffs = __any_sync(FULL_MASK, orBits != toBits && threadIdx.x < NEXT_LOWER);
            bool nextWasUINT_MAX = false;
            if (toNext == ULLONG_MAX && fromNext != ULLONG_MAX)
            {
                toNext = incEdgeCouter(COPY);
                nextWasUINT_MAX = true;
            }

            // each thread gets a value that will be written back to memory
            uint val = thread_load_val(orBits, toNext);
            if (val != toBits)
                __memory__[toIndex + threadIdx.x] = val;

            // as we are merging into copy,
            // we need to also merge the underlying pts sets
            // we do this by running collectBitvectorTargets
            // and then merge thos pts edges again at the end of this loop
            if (applyCopy && diffs)
            {
                uint diffBits = fromBits & ~toBits;
                collectBitvectorTargets<PTS, PTS_NEXT>(to, diffBits, fromBase, storage, numFrom);
            }

            // if no more bitvectors in origin, end loop
            if (fromNext == ULLONG_MAX)
            {
                break;
            }

            // else load next bits
            // keep in mind that we do not use insertBitvector
            // since we need to also merge pts edges
            // instead make toBits undefined manually
            // handle this in toBase > fromBase
            toIndex = toNext;
            if (nextWasUINT_MAX)
            {
                toBits = UINT_MAX;
                toBase = UINT_MAX;
                toNext = ULLONG_MAX;
            }
            else
            {
                toBits = __memory__[toIndex + threadIdx.x];
                toBase = __shfl_sync(FULL_MASK, toBits, BASE);
                toNext = thread_load_size_t(toBits);
            }

            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(FULL_MASK, fromBits, BASE);
            fromNext = __shfl_sync(FULL_MASK, fromBits, NEXT_LOWER);
            fromNext = thread_load_size_t(fromBits);
        }
        else if (toBase < fromBase)
        {
            // if toNext is undefined, we need to allocate a new element
            // after that, we can simply insert the origin bitvector
            if (toNext == ULLONG_MAX)
            {
                index_t newNext = incEdgeCouter(COPY);
                store_size_t(__memory__, toIndex, newNext);
                toIndex = newNext;
                toBits = UINT_MAX;
                toBase = UINT_MAX;
            }
            else
            {
                toIndex = toNext;

                toBits = __memory__[toNext + threadIdx.x];
                toBase = __shfl_sync(FULL_MASK, toBits, BASE);
                toNext = thread_load_size_t(toBits);
            }
        }
        else if (toBase > fromBase)
        {
            // compared to mergeBitvectorPts
            // we need to handle the toBase == UINT_MAX case here
            if (toBase == ULLONG_MAX)
            {
                newVal = fromNext == ULLONG_MAX ? ULLONG_MAX : incEdgeCouter(COPY);
            }
            else
            {
                newVal = incEdgeCouter(COPY);
                // write the current bits from the target element to a new location
                __memory__[newVal + threadIdx.x] = toBits;
            }

            // overwrite the current bits with fromBits (insert before node)
            fromBits = thread_load_val(fromBits, newVal);
            __memory__[toIndex + threadIdx.x] = fromBits;
            if (applyCopy)
            {
                // collect pts edges for resolving the copy edges later
                collectBitvectorTargets<PTS, PTS_NEXT>(to, fromBits, fromBase, storage, numFrom);
            }

            // if next from element is defined, update the bits
            // if not, break for this element
            if (fromNext == ULLONG_MAX)
            {
                break;
            }

            toIndex = newVal;

            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(FULL_MASK, fromBits, BASE);
            fromNext = thread_load_size_t(fromBits);
        }
    }

    // merge collected pts edges
    if (applyCopy && numFrom)
    {
        mergeBitvectors<PTS, PTS_NEXT>(to, numFrom, storage);
    }
}

/**
 *
 * if to BV is empty, just write from BV w/o merging
 * simply assign new elements for next elements
 *
 * \param toIndex target node
 * \param fromBits bits to be written
 * \param fromNext next from index, passed so it does not have to be recomputed
 * \param toRel target relation
 *
 */
__device__ void insertBitvector(index_t toIndex, uint fromBits, index_t fromNext, const uint toRel)
{
    while (1)
    {
        // check if a new bitvector is required
        // if that is the case, allocate a new index for a new element
        index_t newIndex = fromNext == ULLONG_MAX ? ULLONG_MAX : incEdgeCouter(toRel);
        // handle the special next entry, since we can not reuse the fromNext bits
        uint val = thread_load_val(fromBits, newIndex);
        // write new values to target memory location
        __memory__[toIndex + threadIdx.x] = val;

        // exit if no more elements in from bitvector
        if (fromNext == ULLONG_MAX)
            break;

        // start next iteration
        toIndex = newIndex;
        fromBits = __memory__[fromNext + threadIdx.x];
        // use warp intrinsics to get next index in from memory
        fromNext = thread_load_size_t(fromBits);
    }
}

/**
 *
 * merges two bitvectors w/o copy targets,
 * so no recursive calls
 *
 * \param to target node
 * \param fromIndex index of second rel node
 * \param toRel target relation, used for new element allocations
 *
 */
__device__ void mergeBitvectorPts(uint to, index_t fromIndex, const uint toRel)
{
    index_t toIndex = getIndex(to, toRel);
    // read dst out edges
    uint fromBits = __memory__[fromIndex + threadIdx.x];
    // get the base from thread
    uint fromBase = __shfl_sync(FULL_MASK, fromBits, BASE);
    // terminate if no data in from bitvector
    if (fromBase == UINT_MAX)
        return;
    // get the next index from thread
    index_t fromNext = thread_load_size_t(fromBits);

    // share needed data for to indices
    uint toBits = __memory__[toIndex + threadIdx.x];
    uint toBase = __shfl_sync(FULL_MASK, toBits, BASE);
    index_t toNext = thread_load_size_t(toBits);

    if (toBase == UINT_MAX)
    {
        insertBitvector(toIndex, fromBits, fromNext, toRel);
        return;
    }

    while (1)
    {
        if (toBase == fromBase)
        {
            // if target next is undefined, create new edge for more edges
            index_t newToNext = (toNext == ULLONG_MAX && fromNext != ULLONG_MAX) ? incEdgeCouter(toRel) : toNext;
            // union the bits, adding the new edges
            uint orBits = fromBits | toBits;
            // each thread gets a value that will be written back to memory
            uint val = thread_load_val(orBits, newToNext);
            if (val != toBits)
                __memory__[toIndex + threadIdx.x] = val;

            // if no more bitvectors in origin, end loop
            if (fromNext == ULLONG_MAX)
                return;

            // else load next bits
            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(FULL_MASK, fromBits, BASE);
            fromNext = thread_load_size_t(fromBits);
            if (toNext == ULLONG_MAX)
            {
                insertBitvector(newToNext, fromBits, fromNext, toRel);
                return;
            }
            toIndex = newToNext;
            toBits = __memory__[toNext + threadIdx.x];
            toBase = __shfl_sync(FULL_MASK, toBits, BASE);
            toNext = thread_load_size_t(toBits);
        }
        else if (toBase < fromBase)
        {
            // if toNext is undefined, we need to allocate a new element
            // after that, we can simply insert the origin bitvector
            if (toNext == ULLONG_MAX)
            {
                toNext = incEdgeCouter(toRel);
                store_size_t(__memory__, toIndex, toNext);
                insertBitvector(toNext, fromBits, fromNext, toRel);
                return;
            }
            // if toNext is defined, load those to bits for the next iteration
            toIndex = toNext;
            toBits = __memory__[toNext + threadIdx.x];
            toBase = __shfl_sync(FULL_MASK, toBits, BASE);
            toNext = thread_load_size_t(toBits);
        }
        else if (toBase > fromBase)
        {
            // if toBase is greater than frombase
            // we need to insert another bitvector element before toindex
            // and shift the current element back (ref. linked lists)
            index_t newIndex = incEdgeCouter(toRel);
            // write the current bits from the target element to a new location
            __memory__[newIndex + threadIdx.x] = toBits;
            // then overwrite the current bits with fromBits (insert before node)
            uint val = thread_load_val(fromBits, newIndex);
            __memory__[toIndex + threadIdx.x] = val;

            // if next from element is defined, update the bits
            // if not, break for this element
            if (fromNext == ULLONG_MAX)
                return;

            toIndex = newIndex;

            fromBits = __memory__[fromNext + threadIdx.x];
            fromBase = __shfl_sync(FULL_MASK, fromBits, BASE);
            fromNext = thread_load_size_t(fromBits);
        }
    }
}

/**
 *
 * general mergeBitvector function
 * calls mergeBitvectorPts or mergeBitvectorCopy
 * depending on template relations
 *
 * \param to target node
 * \param numDstNodes size of shared memory
 * \param _shared_ shared memory shard containing second rel nodes
 *
 */
template <uint fromRel, uint toRel>
__device__ void mergeBitvectors(const uint to, const uint numDstNodes, uint *_shared_)
{
    // go through all dst nodes, and union the out edges of that node w/ src's out nodes
    for (size_t i = 0; i < numDstNodes; i++)
    {
        index_t fromIndex = getIndex(_shared_[i], fromRel);

        if (toRel == COPY)
        {
            mergeBitvectorCopy(to, fromIndex, _shared_ + 128);
        }
        else
        {
            mergeBitvectorPts(to, fromIndex, toRel);
        }
    }
}

/**
 *
 * read bits and read bitvector uints to calculate
 * and collect target nodes in shared memory
 *
 * \param to target node
 * \param bits bits to be read
 * \param base base of element to be read
 * \param storage shared memory to be used for collection
 * \param usedStorage used to keep track of nodes stored in shared memory
 *
 */
template <uint fromRel, uint toRel>
__device__ void collectBitvectorTargets(const uint to, const uint bits, const uint base, uint *storage, uint &usedStorage)
{
    // create mask for threads w/ dst nodes, except last
    uint nonEmptyThreads = __ballot_sync(FULL_MASK, bits) & BV_THREADS_MASK;
    const uint threadMask = 1 << threadIdx.x;
    const uint myMask = threadMask - 1;
    while (nonEmptyThreads)
    {
        // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
        int leastThread = __ffs(nonEmptyThreads) - 1;
        // remove lsb from nonEmptyThreads (iteration step)
        nonEmptyThreads &= (nonEmptyThreads - 1);
        // share current bits with all threads in warp
        uint current_bits = __shfl_sync(FULL_MASK, bits, leastThread);

        // use the base and the word of the current thread's bits to calculate the target dst id
        uint var = getDstNode(base, leastThread, threadIdx.x);
        // check if this thread is looking at a dst node
        // uint bitActive = (var != 1U) && (current_bits & threadMask);
        uint bitActive = (current_bits & threadMask);
        // count threads that are looking at dst nodes
        uint threadsWithDstNode = __ballot_sync(FULL_MASK, bitActive);
        uint numDstNodes = __popc(threadsWithDstNode);
        if (usedStorage + numDstNodes > 128)
        {
            if (toRel == STORE)
                insert_store_map(to, storage, usedStorage);
            else
                mergeBitvectors<fromRel, toRel>(to, usedStorage, storage);
            usedStorage = 0;
        }
        // calculate pos in shared mem, by counting prev threads that had a dst node
        uint pos = usedStorage + __popc(threadsWithDstNode & myMask);
        if (bitActive)
        {
            storage[pos] = var;
        }
        usedStorage += numDstNodes;
    }
}

/**
 *
 * use the kv store w/ store and pts pairs
 * to add all copy edges resulting from store -> pts paths
 *
 */
__global__ void
__launch_bounds__(THREADS_PER_BLOCK)
    kernel_store2copy()
{
    extern __shared__ uint _sh_[];
    uint *const _shared_ = &_sh_[threadIdx.y * 256];
    for (uint i = blockIdx.x * blockDim.y + threadIdx.y; i < __numKeys__ - 1; i += blockDim.y * gridDim.x)
    {
        uint idx = __offsets__[i];
        uint idx_next = __offsets__[i + 1];

        // load the pts target, this should not change for the next totalDstNodes
        uint pts_target = __key__[idx];

        for (uint j = idx; j < idx_next; j += 32)
        {
            uint numDstNodes = min(idx_next - j, 32U);
            if (j + threadIdx.x < idx_next)
            {
                _shared_[threadIdx.x] = __val__[j + threadIdx.x];
            }
            mergeBitvectors<STORE, COPY>(pts_target, numDstNodes, _shared_);
        }
    }
}

/**
 *
 * kernel_insert_edges to add all edges in kv store to graph
 *
 * \param rel since all edges are coalesced by type we need to know the edge relation
 *
 */
__global__ void kernel_insert_edges(uint rel)
{
    uint index = blockIdx.x * blockDim.y + threadIdx.y;
    uint stride = blockDim.y * gridDim.x;
    uint src, dst, offset, offset_next, j;
    for (int i = index; i < __numKeys__ - 1; i += stride)
    {

        offset = __offsets__[i];
        offset_next = __offsets__[i + 1];
        src = __key__[offset];

        for (j = offset; j < offset_next; j++)
        {
            dst = __val__[j];
            insertEdgeDevice(src, dst, rel);
        }
    }
}

/**
 *
 * kernelWrapper to automatically determine optimal parameters for the kernel
 * as well as synchronizing needed events
 *
 * \param kernel the kernel to be executed
 * \param statusString string to be printed before execution of the kernel
 * \param args **args holds the kernel parameters
 *
 */
__host__ void kernelWrapper(void *kernel, const char *statusString, void **args = 0)
{
    printf("%s", statusString);
    KernelInfo *config = &kernelParameters[kernel];

    if (!config->initialized)
    {
        hipEventCreate(&config->start);
        hipEventCreate(&config->stop);
        int optimalBlockSize;
        int optimalGridSize;

        size_t dynamicSMemUsage = 32 * 256 * sizeof(uint);

        checkCuda(hipOccupancyMaxPotentialBlockSize(&optimalGridSize, &optimalBlockSize, kernel, dynamicSMemUsage, 0));

        printf("[automatic kernel configuration] calculated blkSize: %i and grdSize: %i for kernel [%p]\n", optimalBlockSize, optimalGridSize, kernel);

        dim3 gridSize(optimalGridSize);
        dim3 blockSize(WARP_SIZE, optimalBlockSize / WARP_SIZE);

        config->gridSize = gridSize;
        config->blockSize = blockSize;
        config->initialized = true;
        config->sharedMemory = dynamicSMemUsage;
    }
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipEventRecord(config->start, 0));
    checkCuda(hipLaunchKernel(reinterpret_cast<const void*>(kernel), config->gridSize, config->blockSize, args, config->sharedMemory, 0));
    checkCuda(hipEventRecord(config->stop, 0));
    checkCuda(hipEventSynchronize(config->stop));
    float elapsedTime;
    checkCuda(hipEventElapsedTime(&elapsedTime, config->start, config->stop));
    config->elapsedTime += elapsedTime;
}

/**
 *
 * host method to write edges into the memory
 * stored edges in kv store on gpu to efficiently
 * insert edges
 *
 * \param edges an edgeSet containing all the edges as two vectors
 * \param inv bool whether or not the edges are inverted or not
 * \param rel target relation for the edges
 *
 */
__host__ void insertEdges(edgeSet *edges, int inv, int rel)
{
    uint numEdges = edges->second.size();
    uint N = numEdges + 1;

    assert(N <= KV_SIZE);

    if (inv)
    {
        memcpy(__key__, edges->second.data(), numEdges * sizeof(unsigned int));
        memcpy(__val__, edges->first.data(), numEdges * sizeof(unsigned int));
    }
    else
    {
        memcpy(__key__, edges->first.data(), numEdges * sizeof(unsigned int));
        memcpy(__val__, edges->second.data(), numEdges * sizeof(unsigned int));
    }

    __key__[numEdges] = UINT_MAX;
    __val__[numEdges] = UINT_MAX;

    auto kv_start = thrust::make_zip_iterator(thrust::make_tuple(__key__, __val__));
    thrust::sort(thrust::device, kv_start, kv_start + N);
    __numKeys__ = thrust::unique_by_key_copy(thrust::device, __key__, __key__ + N, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), __offsets__).second - __offsets__;

    dim3 numBlocks(N_BLOCKS);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);
    checkCuda(hipDeviceSynchronize());
    kernel_insert_edges<<<numBlocks, threadsPerBlock, 0>>>(rel);
    checkCuda(hipDeviceSynchronize());
}

/**
 *
 * collect pts targets for src in memory
 * this is used from the host
 *
 * \param src nodeid to collect targets for
 * \param memory memory to probe for targets
 * \param pts vector where to add the collected nodes (reference)
 * \param rel relation to probe in memory
 *
 */
__host__ void collectFromBitvector(uint src, uint *memory, std::vector<uint> &pts, uint rel)
{
    index_t index = getIndex(src, rel);
    while (index != ULLONG_MAX)
    {
        uint base = memory[index + BASE];
        index_t next = load_size_t(memory[index + NEXT_LOWER], memory[index + NEXT_UPPER]);
        if (base == UINT_MAX)
        {
            break;
        }
        for (size_t j = 0; j < BASE; j++)
        {
            uint value = memory[index + j];
            for (size_t k = 0; k < 32; k++)
            {
                if (value & 1)
                {
                    pts.push_back(base * ELEMENT_CARDINALITY + j * 32 + k);
                }
                value >>= 1;
            }
        }
        index = next;
    }
}

/**
 *
 * check of two nodes alias, based on gpu (managed) memory
 *
 * \param a node 1
 * \param a node 2
 * \param memory memory to probe for targets
 *
 * \return bool for alias relation
 *
 */
__host__ bool aliasBV(uint a, uint b, uint *memory)
{
    std::vector<uint> ptsA, ptsB;

    collectFromBitvector(a, memory, ptsA, PTS);
    collectFromBitvector(b, memory, ptsB, PTS);

    for (uint target : ptsA)
        if (std::find(ptsB.begin(), ptsB.end(), target) != ptsB.end())
            return true;

    return false;
}

/**
 *
 * same as insertBitvector, but link instead of copying
 * used when updating pts.
 * reuse pts memory for currpts
 *
 * \param var currently updating this node
 * \param ptsIndex index in the pts, dest for data
 * \param currDiffPtsIndex index in currpts, 2nd dest for data, next to nextpts index
 * \param diffPtsBits nextpts bits, source of data
 * \param diffPtsNext nextpts next, rest of source data
 *
 */
__device__ void insertBitvectorAndLink(uint var, const index_t ptsIndex, index_t &currDiffPtsIndex, const uint diffPtsBits, const index_t diffPtsNext)
{
    insertBitvector(ptsIndex, diffPtsBits, diffPtsNext, PTS);
    if (currDiffPtsIndex != ULLONG_MAX)
    {
        store_size_t(__memory__, currDiffPtsIndex, ptsIndex);
    }
    else
    {
        currDiffPtsIndex = getIndex(var, PTS_CURR);
        uint ptsBits = __memory__[ptsIndex + threadIdx.x];
        __memory__[currDiffPtsIndex + threadIdx.x] = ptsBits;
    }
}

/**
 * Update the current, next and total PTS sets of a variable. In the last iteration of the main
 * loop, points-to edges have been added to NEXT_DIFF_PTS. However, many of them might already be
 * present in PTS. The purpose of this function is to update PTS as PTS U NEXT_DIFF_PTS, and set
 * PTS_CURR as the difference between the old and new PTS for the given variable.
 *
 * @param var nodeid for currently updating variable
 * @return true if new pts edges have been added to this variable
 */
__device__ bool computeDiffPts(const uint var)
{
    // get diffpts index
    const index_t diffPtsIndex = getIndex(var, PTS_NEXT);

    // read diffpts data
    uint diffPtsBits = __memory__[diffPtsIndex + threadIdx.x];
    uint diffPtsBase = __shfl_sync(FULL_MASK, diffPtsBits, BASE);

    if (diffPtsBase == UINT_MAX)
    {
        return false;
    }
    // get next element for diffpts
    index_t diffPtsNext = thread_load_size_t(diffPtsBits);
    // reset the diffpts data
    __memory__[diffPtsIndex + threadIdx.x] = UINT_MAX;

    // get pts index
    index_t ptsIndex = getIndex(var, PTS);

    // get pts data
    uint ptsBits = __memory__[ptsIndex + threadIdx.x];
    uint ptsBase = __shfl_sync(FULL_MASK, ptsBits, BASE);

    if (ptsBase == UINT_MAX)
    {
        // use dummy variable for currDiffPtsIndex and insert
        index_t tmp = ULLONG_MAX;
        insertBitvectorAndLink(var, ptsIndex, tmp, diffPtsBits, diffPtsNext);
        return true;
    }
    // get next pts element
    index_t ptsNext = thread_load_size_t(ptsBits);

    // init currDiffPtsIndex to undef
    index_t currDiffPtsIndex = ULLONG_MAX;
    while (1)
    {
        if (ptsBase > diffPtsBase)
        {
            // insert new element for diffpts data
            // and write previous pts data to new element
            index_t newIndex = incEdgeCouter(PTS);
            __memory__[newIndex + threadIdx.x] = ptsBits;
            uint val = thread_load_val(diffPtsBits, newIndex);
            __memory__[ptsIndex + threadIdx.x] = val;

            // update pts index
            ptsIndex = newIndex;

            // also write to currpts, instead of only writing to pts
            newIndex = currDiffPtsIndex == ULLONG_MAX ? getIndex(var, PTS_CURR) : incEdgeCouter(PTS_CURR);
            val = threadIdx.x < NEXT_LOWER ? diffPtsBits : UINT_MAX;
            __memory__[newIndex + threadIdx.x] = val;
            if (currDiffPtsIndex != ULLONG_MAX)
                store_size_t(__memory__, currDiffPtsIndex, newIndex);

            // abort if diffpts next is undefined alse update index
            if (diffPtsNext == ULLONG_MAX)
                return true;
            currDiffPtsIndex = newIndex;

            // get next diffpts data
            diffPtsBits = __memory__[diffPtsNext + threadIdx.x];
            diffPtsBase = __shfl_sync(FULL_MASK, diffPtsBits, BASE);
            diffPtsNext = thread_load_size_t(diffPtsBits);
        }
        else if (ptsBase == diffPtsBase)
        {
            // calculate bits that should be merged w/ pts
            index_t newPtsNext = (ptsNext == ULLONG_MAX && diffPtsNext != ULLONG_MAX) ? incEdgeCouter(PTS) : ptsNext;
            uint orBits = thread_load_val(ptsBits | diffPtsBits, newPtsNext);
            uint ballot = __ballot_sync(FULL_MASK, orBits != ptsBits);
            if (ballot)
            {
                // write the orbits
                __memory__[ptsIndex + threadIdx.x] = orBits;
                if (ballot & ((1 << BASE) - 1))
                {
                    orBits = diffPtsBits & ~ptsBits;
                    if (threadIdx.x == BASE)
                    {
                        orBits = ptsBase;
                    }
                    else if (threadIdx.x == NEXT_LOWER)
                    {
                        orBits = UINT_MAX;
                    }
                    else if (threadIdx.x == NEXT_UPPER)
                    {
                        orBits = UINT_MAX;
                    }

                    // now write diffPtsBits & ~ptsBits to currpts at correct index
                    index_t newIndex;
                    if (currDiffPtsIndex != ULLONG_MAX)
                    {

                        newIndex = incEdgeCouter(PTS_CURR);
                        store_size_t(__memory__, currDiffPtsIndex, newIndex);
                    }
                    else
                    {
                        newIndex = getIndex(var, PTS_CURR);
                    }
                    __memory__[newIndex + threadIdx.x] = orBits;
                    currDiffPtsIndex = newIndex;
                }
            }

            // abort of diffnext in undefined
            if (diffPtsNext == ULLONG_MAX)
            {
                return (currDiffPtsIndex != ULLONG_MAX);
            }

            // else get next diff data
            diffPtsBits = __memory__[diffPtsNext + threadIdx.x];
            diffPtsBase = __shfl_sync(FULL_MASK, diffPtsBits, BASE);
            diffPtsNext = thread_load_size_t(diffPtsBits);

            // if pts next is undefined skip next iteration and insertBitvectorAndLink instead
            if (ptsNext == ULLONG_MAX)
            {
                insertBitvectorAndLink(var, newPtsNext, currDiffPtsIndex, diffPtsBits, diffPtsNext);
                return true;
            }

            // else iterate and get next pts data as well
            ptsIndex = ptsNext;

            ptsBits = __memory__[ptsIndex + threadIdx.x];
            ptsBase = __shfl_sync(FULL_MASK, ptsBits, BASE);
            ptsNext = thread_load_size_t(ptsBits);
        }
        else if (ptsBase < diffPtsBase)
        {
            // when ptsBase is too small and has no next, insertBitvectorAndLink
            if (ptsNext == ULLONG_MAX)
            {
                index_t newPtsIndex = incEdgeCouter(PTS);
                store_size_t(__memory__, ptsIndex, newPtsIndex);
                insertBitvectorAndLink(var, newPtsIndex, currDiffPtsIndex, diffPtsBits, diffPtsNext);
                return true;
            }

            // else get next pts data and iterate
            ptsIndex = ptsNext;
            ptsBits = __memory__[ptsIndex + threadIdx.x];
            ptsBase = __shfl_sync(FULL_MASK, ptsBits, BASE);
            ptsNext = thread_load_size_t(ptsBits);
        }
    }
}

/**
 * debug function to do some sanity checks on the memory
 * i.e. check if index == next, which would result in infinite loops
 *
 */
__global__ void kernel_memoryCheck()
{
    __syncthreads();

    uint start = blockIdx.x * blockDim.y + threadIdx.y;
    uint stride = blockDim.y * gridDim.x;
    uint bits, base;
    index_t next, index;

    for (int i = start; i < V; i += stride)
    {
        index = getIndex(i, PTS_CURR);
        while (index != ULLONG_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(FULL_MASK, bits, BASE);
            if (base == UINT_MAX)
                break;

            next = __shfl_sync(FULL_MASK, bits, NEXT_LOWER);
            if (!threadIdx.x && next == index)
            {
                printf("currpts index: %llu has smaller next: %llu, freeList: %llu\n", index, next, __freeList__[PTS_CURR]);
                break;
            }
            index = next;
        }
    }
    __syncthreads();

    for (int i = start; i < V; i += stride)
    {
        index = getIndex(i, PTS);
        while (index != ULLONG_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(FULL_MASK, bits, BASE);
            if (base == UINT_MAX)
                break;

            next = __shfl_sync(FULL_MASK, bits, NEXT_LOWER);
            if (!threadIdx.x && next == index)
            {
                printf("pts index: %llu has smaller next: %llu, freeList: %llu\n", index, next, __freeList__[PTS]);
                break;
            }
            index = next;
        }
    }
    __syncthreads();

    for (int i = start; i < V; i += stride)
    {
        index = getIndex(i, PTS_NEXT);
        while (index != ULLONG_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(FULL_MASK, bits, BASE);
            if (base == UINT_MAX)
                break;

            next = __shfl_sync(FULL_MASK, bits, NEXT_LOWER);
            if (!threadIdx.x && next == index)
            {
                printf("nextpts index: %llu has smaller next: %llu, freeList: %llu\n", index, next, __freeList__[PTS_NEXT]);
                break;
            }
            index = next;
        }
    }
    __syncthreads();
}

/**
 *
 * count targets in type of bitvector
 * does not have to be pts, can be any memory region containing bitvectors
 *
 * \param rel for which to count the targets
 *
 */
__global__ void kernel_count_pts(uint rel)
{
    uint start = blockIdx.x * blockDim.y + threadIdx.y;
    uint stride = blockDim.y * gridDim.x;
    uint bits, base;
    index_t next, index;

    for (int i = start; i < V; i += stride)
    {
        index = getIndex(i, rel);
        while (index != ULLONG_MAX)
        {
            bits = __memory__[index + threadIdx.x];
            base = __shfl_sync(FULL_MASK, bits, BASE);
            if (base == UINT_MAX)
                break;

            uint value = threadIdx.x < BASE ? __popc(bits) : 0;

#if __CUDA_ARCH__ >= 800 && 0
            __reduce_add_sync(FULL_MASK, value);
#else
            for (int i = 16; i >= 1; i /= 2)
                value += __shfl_xor_sync(FULL_MASK, value, i);
#endif

            if (!threadIdx.x)
                atomicAdd(&__counter__, value);

            next = thread_load_size_t(bits);

            index = next;
        }
    }
}

/**
 *
 * update the pointer memory regions
 * PTS = PTS U PTS_NEXT
 * PTS_CURR = PTS_NEXT \ PTS
 *
 */
__global__ void kernel_updatePts()
{
    __done__ = true;
    bool newWork = false;
    for (uint i = blockIdx.x * blockDim.y + threadIdx.y; i < V; i += blockDim.y * gridDim.x)
    {
        bool newStuff = computeDiffPts(i);
        newWork |= newStuff;
        if (!newStuff)
        {
            const index_t currPtsHeadIndex = getIndex(i, PTS_CURR);
            __memory__[currPtsHeadIndex + threadIdx.x] = UINT_MAX;
        }
    }
    if (newWork)
    {
        __done__ = false;
    }
    if (resetWorklistIndex())
    {
        tmpFreePtsCurr = __freeList__[PTS_CURR];
        __freeList__[PTS_CURR] = OFFSET_PTS_CURR + __reservedHeader__;
        __freeList__[PTS_NEXT] = OFFSET_PTS_NEXT + __reservedHeader__;
    }
}

/**
 *
 * general rewrite rule
 * given two relations, insert new edges for a third relation connecting both
 * X -a-> Y -b-> Z  => X -c-> Z
 *
 * \param src apply rewrite rule for this variable
 * \param _shared_ block of allocated shared memory
 *
 */
template <uint originRel, uint fromRel, uint toRel>
__device__ void rewriteRule(const uint src, uint *const _shared_)
{
    uint usedShared = 0;
    index_t index = getIndex(src, originRel);
    do
    {
        uint bits = __memory__[index + threadIdx.x];
        uint base = __shfl_sync(FULL_MASK, bits, BASE);
        if (base == UINT_MAX)
            break;
        index = thread_load_size_t(bits);
        collectBitvectorTargets<fromRel, toRel>(src, bits, base, _shared_, usedShared);
    } while (index != ULLONG_MAX);
    if (usedShared)
    {
        if (fromRel == STORE)
            insert_store_map(src, _shared_, usedShared);
        else
            mergeBitvectors<fromRel, toRel>(src, usedShared, _shared_);
    }
}

/**
 *
 * this is the main kernel
 * which applies copy load and partly store rewrite rules
 *
 */
__global__ void
__launch_bounds__(THREADS_PER_BLOCK)
    kernel()
{
    extern __shared__ uint _sh_[];
    uint *const _shared_ = &_sh_[threadIdx.y * 256];
    uint to = V;
    uint src = getAndIncrement(&__worklistIndex1__, 1);
    while (src < to)
    {
        rewriteRule<COPY, PTS_CURR, PTS_NEXT>(src, _shared_ + 128);
        rewriteRule<LOAD, PTS_CURR, COPY>(src, _shared_);

        src = getAndIncrement(&__worklistIndex1__, 1);
    }
    to = __numStoreConstraints__;
    src = getAndIncrement(&__worklistIndex0__, 1);
    while (src < to)
    {
        src = __storeConstraints__[src];
        if (src != UINT_MAX)
        {
            rewriteRule<PTS_CURR, STORE, STORE>(src, _shared_);
        }
        src = getAndIncrement(&__worklistIndex0__, 1);
    }
    if (resetWorklistIndex())
    {
        __key__[__storeMapHead__] = UINT_MAX;
        __val__[__storeMapHead__] = UINT_MAX;
        __numKeys__ = __storeMapHead__ + 1;
        __storeMapHead__ = 0;
        __worklistIndex1__ = 0;
    }
}

/**
 *
 * helper function to print a bitvector element in its binary representation
 * very useful for debugging
 *
 * \param memory graph to print from
 * \param src nodeid / memory location to read
 * \param rel target relation to read from memory
 * \param isNodeId whether to calculate index from src or use as is, first is the default behaviour
 *
 */
__host__ void printWord(uint *memory, index_t src, uint rel, bool isNodeId = true)
{
    index_t start;
    if (isNodeId)
        start = getIndex(src, rel);
    else
        start = src;

    for (size_t i = 0; i < 32; i++)
    {
        uint checkpoint = memory[start + i];
        std::cout << checkpoint << "\n";
        std::bitset<sizeof(uint) * 8> x(checkpoint);
        std::cout << x << '\n';
    }
    std::cout << '\n';
}

/**
 *
 * helper function to print all pts for a relation in a graph
 *
 * \param V number of nodes in graph
 * \param memory graph to print from
 * \param rel target relation to read from memory
 *
 */
__host__ void printAllPts(uint V, uint *memory, uint rel)
{
    for (size_t i = 0; i < V; i++)
    {
        uint index = getIndex(i, rel);
        printf("\n %lu -> [", i);
        while (index != UINT_MAX)
        {
            uint base = __memory__[index + BASE];
            uint next = __memory__[index + NEXT_LOWER];
            if (base == UINT_MAX)
            {
                break;
            }
            for (size_t j = 0; j < BASE; j++)
            {
                uint value = __memory__[index + j];
                for (size_t k = 0; k < 32; k++)
                {
                    if (value & 1)
                    {
                        printf("%u ", getDstNode(base, j, k));
                    }
                    value >>= 1;
                }
            }
            index = next;
        }
        printf("]");
    }
}

/**
 *
 * helper function to print all pts minimally for a relation in a graph
 * minimal means that empty sets are omitted for easier reading / parsing
 *
 * \param V number of nodes in graph
 * \param memory graph to print from
 * \param rel target relation to read from memory
 *
 */
__host__ void printAllPtsMinimal(uint V, uint *memory, uint rel)
{
    for (size_t i = 0; i < V; i++)
    {
        std::vector<uint> targets;
        collectFromBitvector(i, memory, targets, rel);
        if (targets.size())
        {
            printf("\n %lu -> [", i);
            for (auto t : targets)
                printf("%u ", t);
            printf("]");
        }
    }
}

/**
 *
 * print the memory state for a single relation in memory
 * use freelist ot get state
 *
 * \param start start offset of the memory region (rel)
 * \param end end offset of the memory region (rel)
 * \param rel target relation to read from freelist
 *
 */
__host__ void printMemory(index_t start, index_t end, uint rel)
{
    index_t usedUints;
    if (rel == PTS_CURR)
        usedUints = tmpFreePtsCurr - start;
    else
        usedUints = __freeList__[rel] - start;
    size_t usedBytes = usedUints * sizeof(uint);
    size_t totalBytes = (end - start) * sizeof(uint);
    assert(usedBytes < totalBytes);
    printf("%12s Elements:(uints)%16llu\t[%10.3f MiB / %5lu MiB]\n", relNames[rel], usedUints, (usedBytes / (1024.0 * 1024.0)), totalBytes >> 20);
}

/**
 *
 * helper function to print the current state of memory
 * print used and available memory for all relations in a graph
 *
 */
__host__ void reportMemory()
{
    printf("##### MEMORY USAGE\n");
    printMemory(OFFSET_PTS, TOTAL_MEMORY_LENGTH, PTS);
    printMemory(OFFSET_PTS_CURR, OFFSET_PTS, PTS_CURR);
    printMemory(OFFSET_PTS_NEXT, OFFSET_PTS_CURR, PTS_NEXT);
    printMemory(OFFSET_COPY, OFFSET_PTS_NEXT, COPY);
    printMemory(OFFSET_LOAD, OFFSET_COPY, LOAD);
    printMemory(OFFSET_STORE, OFFSET_LOAD, STORE);
    printf("##### MEMORY USAGE\n");
}

/**
 *
 * mainloop to calculate transitive closure for a given andersen analysis problem
 * \return pointer for final memory, containing all relations
 *
 */
__host__ uint *run(unsigned int numNodes, edgeSet *addrEdges, edgeSet *directEdges, edgeSet *loadEdges, edgeSet *storeEdges, std::function<uint(uint *, edgeSet *pts, edgeSet *copy)> callgraphCallback)
{
    setlocale(LC_NUMERIC, "");

    int N_GPU;
    checkCuda(hipGetDeviceCount(&N_GPU));
    hipStream_t streams[N_GPU];

    for (int i = 0; i < N_GPU; i++)
    {
        checkCuda(hipSetDevice(i));
        checkCuda(hipFree(0));
        checkCuda(hipStreamCreate(&streams[i]));
    }
    // hipDeviceProp_t prop; // CUDA device properties variable
    // checkCuda(hipGetDeviceProperties(&prop, 0));
    // printf("total global memory available:\n\t\t%lu\n", prop.totalGlobalMem);
    // printf("total bytes: \t%lu\n", SIZE_TOTAL_BYTES);
    size_t numStoreDst = storeEdges->second.size();
    uint *memory;
    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&memory, SIZE_TOTAL_BYTES));
    // checkCuda(hipHostAlloc(&memory, SIZE_TOTAL_BYTES, hipHostMallocMapped | 0));
    checkCuda(hipMallocManaged(&__key__, KV_SIZE * sizeof(uint)));
    checkCuda(hipMallocManaged(&__val__, KV_SIZE * sizeof(uint)));
    checkCuda(hipMallocManaged(&__offsets__, KV_SIZE * sizeof(uint)));
    checkCuda(hipMallocManaged(&__storeConstraints__, numStoreDst * sizeof(uint)));

    // set all values to UINT_MAX
    hipMemset(memory, UCHAR_MAX, SIZE_TOTAL_BYTES);
    hipMemset(__key__, UCHAR_MAX, KV_SIZE * sizeof(unsigned int));
    hipMemset(__val__, UCHAR_MAX, KV_SIZE * sizeof(unsigned int));
    hipMemset(__offsets__, UCHAR_MAX, KV_SIZE * sizeof(unsigned int));

    // move the store constraints into managed memory and sort / unique
    memcpy(__storeConstraints__, storeEdges->second.data(), numStoreDst * sizeof(uint));
    thrust::sort(__storeConstraints__, __storeConstraints__ + numStoreDst);
    __numStoreConstraints__ = thrust::unique(__storeConstraints__, __storeConstraints__ + numStoreDst) - __storeConstraints__;

    // num of vertices
    V = numNodes;
    size_t V_max = (size_t)ceil(1.2 * V);

    // move managed memory ptrs into device memory
    __memory__ = memory;
    // checkCuda(hipHostGetDevicePointer(&__memory__, memory, 0));

    // reserve 20% for new edges added by gep offsets
    __reservedHeader__ = V_max * ELEMENT_WIDTH;
    __freeList__[PTS] = OFFSET_PTS + __reservedHeader__;
    __freeList__[PTS_CURR] = OFFSET_PTS_CURR + __reservedHeader__;
    __freeList__[PTS_NEXT] = OFFSET_PTS_NEXT + __reservedHeader__;
    __freeList__[COPY] = OFFSET_COPY + __reservedHeader__;
    __freeList__[LOAD] = OFFSET_LOAD + __reservedHeader__;
    __freeList__[STORE] = OFFSET_STORE + __reservedHeader__;

    insertEdges(addrEdges, 1, PTS_NEXT);
    insertEdges(directEdges, 1, COPY);
    insertEdges(loadEdges, 1, LOAD);
    insertEdges(storeEdges, 1, STORE);

    dim3 numBlocks(N_BLOCKS);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);

    size_t iter = 0;

    std::chrono::high_resolution_clock::time_point before, after;
    std::chrono::duration<double, std::milli> timeThrust(0), timeSvf(0), timeUpdate(0), timeKernel(0), timeStore(0);

    while (1)
    {
        ++iter;
        printf("updating info \n");
        checkCuda(hipDeviceSynchronize());
        before = std::chrono::high_resolution_clock::now();
        kernel_updatePts<<<numBlocks, threadsPerBlock, 0>>>();
        checkCuda(hipDeviceSynchronize());
        after = std::chrono::high_resolution_clock::now();
        timeUpdate += std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before);

        checkCuda(hipDeviceSynchronize());
        kernel_count_pts<<<numBlocks, threadsPerBlock, 0>>>(PTS_CURR);
        checkCuda(hipDeviceSynchronize());
        printf("\tnum currpts after update: %'u\n", __counter__);
        __counter__ = 0;

        if (__done__)
        {
            std::cout << "\t\tno change recorded, aborting main loop in iter: " << iter << "\n";
            break;
        }

        edgeSet newPts, newCopys;
        std::future<uint> cbFuture = std::async(callgraphCallback, memory, &newPts, &newCopys);

        printf("\trunning main kernel\n");
        checkCuda(hipDeviceSynchronize());
        before = std::chrono::high_resolution_clock::now();

        for (int i = 0; i < N_GPU; i++)
        {
            hipSetDevice(i);
            kernel<<<numBlocks, threadsPerBlock, 256 * sizeof(uint) * threadsPerBlock.y, streams[i]>>>();
        }
        for (int i = 0; i < N_GPU; i++)
        {
            hipSetDevice(i);
            hipStreamSynchronize(streams[i]);
        }
        hipSetDevice(0);

        // kernel<<<numBlocks, threadsPerBlock, 256 * sizeof(uint) * threadsPerBlock.y>>>();
        // checkCuda(hipDeviceSynchronize());
        after = std::chrono::high_resolution_clock::now();
        timeKernel += std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before);

        before = std::chrono::high_resolution_clock::now();
        std::cout << "\tsorting and calculating offsets for store kernel\n";
        auto sync_exec_policy = thrust::device;
        thrust::zip_iterator<thrust::tuple<uint *, uint *>> kv_start = thrust::make_zip_iterator(thrust::make_tuple(__key__, __val__));
        thrust::sort(sync_exec_policy, kv_start, kv_start + __numKeys__);
        __numKeys__ = thrust::unique_by_key_copy(sync_exec_policy, __key__, __key__ + __numKeys__, thrust::make_counting_iterator(0), thrust::make_discard_iterator(), __offsets__).second - __offsets__;
        after = std::chrono::high_resolution_clock::now();
        timeThrust += std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before);

        checkCuda(hipDeviceSynchronize());
        before = std::chrono::high_resolution_clock::now();
        kernel_store2copy<<<numBlocks, threadsPerBlock, 256 * sizeof(uint) * threadsPerBlock.y>>>();
        checkCuda(hipDeviceSynchronize());
        after = std::chrono::high_resolution_clock::now();
        timeStore += std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before);

        std::cout << "\thandle gep edges & ind calls";
        before = std::chrono::high_resolution_clock::now();
        uint Vnew = cbFuture.get();
        printf("  inserting %lu  ", newPts.first.size());
        insertEdges(&newPts, 0, PTS_NEXT);
        insertEdges(&newCopys, 1, COPY);
        printf("  inserting done  ");
        std::cout << "\tnew nodes: " << Vnew - V << " new V: " << Vnew << "\n";
        after = std::chrono::high_resolution_clock::now();
        timeSvf += std::chrono::duration_cast<std::chrono::duration<double, std::milli>>(after - before);
        V = Vnew;
        reportMemory();
    }

    printf("time update: %.3f ms\n", timeUpdate.count());
    printf("time kernel: %.3f ms\n", timeKernel.count());
    printf("time thrust: %.3f ms\n", timeThrust.count());
    printf("time store : %.3f ms\n", timeStore.count());
    printf("time svf   : %.3f ms\n", timeSvf.count());

    // Free memory
    checkCuda(hipFree(__key__));
    checkCuda(hipFree(__val__));
    checkCuda(hipFree(__offsets__));
    checkCuda(hipFree(__storeConstraints__));

    return memory;
}