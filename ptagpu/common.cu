#include "hip/hip_runtime.h"
#include "common.cuh"
#include <iostream>
#include <bitset>

/**
 * __ptsFreeList__
 * this is the head of the free list
 * keeps track of last allocated memory location
 * access needs to be atomic to prevent collisions
 *
 */
__device__ uint __ptsFreeList__;

/**
 * getHeadIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 *
 * \return index of the
 *
 */
__host__ __device__ size_t getHeadIndex(uint src, uint *graph)
{
    return 0;
}

__device__ uint incEdgeCouter()
{
    __shared__ volatile uint _shared_[THREADS_PER_BLOCK / WARP_SIZE];
    if (threadIdx.x == 0)
    {
        _shared_[threadIdx.y] = atomicAdd(&__ptsFreeList__, 32);
    }
    return _shared_[threadIdx.y];
}

__device__ uint insertEdgeDevice(uint src, uint dst, uint *graph)
{
    uint index = src * 32;
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);
    uint myBits = 0;

    if (threadIdx.x == word)
        myBits = 1 << bit;
    else if (threadIdx.x == BASE)
        myBits = base;
    else if (threadIdx.x == NEXT)
        myBits = UINT_MAX;

    while (1)
    {
        uint toBits = graph[index + threadIdx.x];
        uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
        if (toBase == UINT_MAX)
        {
            graph[index + threadIdx.x] = myBits;
            return index;
        }
        if (toBase == base)
        {
            uint orBits = toBits | myBits;
            if (orBits != toBits && threadIdx.x < NEXT)
                graph[index + threadIdx.x] = orBits;

            return index;
        }
        if (toBase < base)
        {
            uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
            if (toNext == UINT_MAX)
            {
                uint newIndex = incEdgeCouter();
                uint val = threadIdx.x == NEXT ? newIndex : myBits;
                graph[newIndex + threadIdx.x] = val;
                return newIndex;
            }
            index = toNext;
        }
        else
        {
            uint newIndex = incEdgeCouter();
            graph[newIndex + threadIdx.x] = myBits;
            uint val = threadIdx.x == NEXT ? newIndex : myBits;
            graph[index + threadIdx.x] = val;
            return index;
        }
    }
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ void insertEdge(uint src, uint dst, uint *graph)
{
    uint index = src * 32;
    uint base = BASE_OF(dst);
    uint word = WORD_OF(dst);
    uint bit = BIT_OF(dst);

    while (1)
    {

        uint toBits = graph[index + word];
        uint toBase = graph[index + BASE];
        uint toNext = graph[index + NEXT];
    }
    if (graph[index + BASE] == UINT_MAX){
        for (size_t i = 0; i < ELEMENT_WIDTH - 2; i++)
            graph[index + i] = 0;
        graph[index + BASE] = base;
    }

    graph[index + word] |= 1 << bit;
}

__device__ void insertBitvector(uint *originMemory, uint *targetMemory, uint toIndex, uint fromBits)
{
    while (1)
    {
        // use warp intrinsics to get next index in from memory
        uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
        // check if a new bitvector is required
        // if that is the case, allocate a new index for a new element
        uint toNext = fromNext == UINT_MAX ? UINT_MAX : incEdgeCouter();
        // handle the special next entry, since we can not reuse the fromNext bits
        uint val = threadIdx.x == NEXT ? toNext : fromBits;
        // write new values to target memory location
        targetMemory[toIndex + threadIdx.x] = val;
        // exit if no more elements in from bitvector
        if (fromNext == UINT_MAX)
            return;
        toIndex = toNext;
        fromBits = originMemory[fromNext + threadIdx.x];
    }
}

__global__ void kernel(int n, uint *A, uint *B, uint *C)
{
    // each warp gets a shared block for one access to global memory
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 128];
    uint *const _shared_ = &_sh_[threadIdx.y * 128];
    for (uint src = blockIdx.x * blockDim.x + threadIdx.y; src < n; src += blockDim.x * gridDim.x)
    {
        uint index = src * 32;
        do
        {
            uint bits = A[index + threadIdx.x];
            uint base = __shfl_sync(0xFFFFFFFF, bits, 30);
            if (base == UINT_MAX)
                break;
            // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
            uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
            const uint threadMask = 1 << threadIdx.x;
            const uint myMask = threadMask - 1;
            while (nonEmptyThreads)
            {
                // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
                int leastThread = __ffs(nonEmptyThreads) - 1;
                // remove lsb from nonEmptyThreads (iteration step)
                nonEmptyThreads &= (nonEmptyThreads - 1);
                // share current bits with all threads in warp
                uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);

                // use the base and the word of the current thread's bits to calculate the target dst id
                uint var = base * 30 * 32 + 32 * leastThread + threadIdx.x;
                // check if this thread is looking at a dst node
                // uint bitActive = (var != 1U) && (current_bits & threadMask);
                uint bitActive = (current_bits & threadMask);
                // count threads that are looking at dst nodes
                uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
                uint numDstNodes = __popc(threadsWithDstNode);
                // calculate pos in shared mem, by counting prev threads that had a dst node
                uint pos = 0 + __popc(threadsWithDstNode & myMask);
                if (bitActive)
                {
                    _shared_[pos] = var;
                }
                if (numDstNodes)
                {
                    // go through all dst nodes, and union the out edges of that node w/ src's out nodes
                    for (size_t i = 0; i < numDstNodes; i++)
                    {
                        uint fromIndex = _shared_[i] * 32;
                        // read dst out edges
                        uint fromBits = B[fromIndex + threadIdx.x];
                        // get the base from thread nr 30
                        uint fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
                        // terminate if no data in from from bitvector
                        if (fromBase == UINT_MAX)
                            continue;
                        // get the next index from thread nr 31
                        uint fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);

                        // share needed data for to indices
                        uint toIndex = index;
                        uint toBits = C[toIndex + threadIdx.x];
                        uint toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                        uint toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);

                        if (toBase == UINT_MAX)
                            insertBitvector(B, C, toIndex, fromBits);
                        break;
                        while (1)
                        {
                            if (toBase == fromBase)
                            {
                                // if target next is undefined, create new edge for more edges
                                uint newToNext = (toNext == UINT_MAX && fromNext != UINT_MAX) ? incEdgeCouter() : toNext;
                                // union the bits, adding the new edges
                                uint orBits = fromBits | toBits;
                                // each thread gets a value that will be written back to memory
                                uint val = threadIdx.x == NEXT ? newToNext : orBits;
                                if (val != toBits)
                                {
                                    C[toIndex + threadIdx.x] = val;
                                }

                                // if no more bitvectors in origin, end loop
                                if (fromNext == UINT_MAX)
                                {
                                    break;
                                }
                                // else load next bits
                                fromBits = C[fromNext + threadIdx.x];
                                fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
                                fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
                                if (toNext == UINT_MAX)
                                {
                                    insertBitvector(B, C, toIndex, fromBits);
                                    break;
                                }
                                toIndex = newToNext;
                                toBits = C[toNext + threadIdx.x];
                                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
                            }
                            else if (toBase < fromBase)
                            {
                                // if toNext is undefined, we need to allocate a new element
                                // after that, we can simply insert teh origin bitvector
                                if (toNext == UINT_MAX)
                                {
                                    toNext = incEdgeCouter();
                                    insertBitvector(B, C, toNext, fromBits);
                                    break;
                                }
                                // if toNext is defined, load those to bits for the next iteration
                                toIndex = toNext;
                                toBits = C[toNext + threadIdx.x];
                                toBase = __shfl_sync(0xFFFFFFFF, toBits, 30);
                                toNext = __shfl_sync(0xFFFFFFFF, toBits, 31);
                            }
                            else if (toBase > fromBase)
                            {
                                // if toBase is greater than frombase
                                // we need to insert enother bitvector element before toindex
                                // and shift the current element back (ref. linked lists)
                                uint newIndex = incEdgeCouter();
                                // write the current bits from the target element to a new location
                                C[newIndex + threadIdx.x] = toBits;
                                // then overwrite the current bits with fromBits (insert before node)
                                uint val = threadIdx.x == NEXT ? newIndex : fromBits;
                                C[toIndex + threadIdx.x] = val;

                                // if next from element is defined, update the bits
                                if (fromNext == UINT_MAX)
                                    return;

                                toIndex = newIndex;

                                fromBits = C[fromNext + threadIdx.x];
                                fromBase = __shfl_sync(0xFFFFFFFF, fromBits, 30);
                                fromNext = __shfl_sync(0xFFFFFFFF, fromBits, 31);
                            }
                        }
                    }
                }
            }
            index = __shfl_sync(0xFFFFFFFF, bits, 31);
        } while (index != UINT_MAX);
    }
}

__host__ int run()
{
    // CUDA kernel to add elements of two arrays

    int N = 1 << 20;
    uint *pts, *prevPtsDiff, *currPtsDiff, *invCopy, *invStore, *invLoad;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&prevPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&currPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invCopy, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invStore, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invLoad, N * sizeof(uint1)));

    // set all values to UINT_MAX
    hipMemset(pts, UINT_MAX, N);
    hipMemset(prevPtsDiff, UINT_MAX, N);
    hipMemset(currPtsDiff, UINT_MAX, N);
    hipMemset(invCopy, UINT_MAX, N);
    hipMemset(invStore, UINT_MAX, N);
    hipMemset(invLoad, UINT_MAX, N);

    // num of vertices
    size_t V{3};

    // insertEdge(1, 0, pts);
    // insertEdge(2, 1, invLoad);
    // insertEdge(1, 3, invStore);
    // insertEdge(3, 4, pts);
    numElements = V;
    insertEdge(0, 1, invCopy);
    insertEdge(1, 2, pts);

    uint numPtsElementsFree = V * ELEMENT_WIDTH;
    hipMemcpyToSymbol(HIP_SYMBOL(__ptsFreeList__), &numPtsElementsFree, sizeof(uint));

    dim3 numBlocks(16);
    dim3 threadsPerBlock(WARP_SIZE, THREADS_PER_BLOCK / WARP_SIZE);
    kernel<<<numBlocks, threadsPerBlock>>>(V, invCopy, pts, pts);

    // Wait for GPU to finish before accessing on host
    checkCuda(hipDeviceSynchronize());

    // Free memory
    checkCuda(hipFree(pts));
    checkCuda(hipFree(prevPtsDiff));
    checkCuda(hipFree(currPtsDiff));
    checkCuda(hipFree(invCopy));
    checkCuda(hipFree(invStore));
    checkCuda(hipFree(invLoad));

    return 0;
}