#include "hip/hip_runtime.h"
#include "common.cuh"

/**
 * getHeadIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 *
 * \return index of the
 *
 */
__host__ __device__ size_t getHeadIndex(uint src, uint *graph)
{
    return 0;
}

/**
 * Basic function to insert edges into graph
 * This function is slow and running on the CPU
 * it is also assumed, that all edges have the same base and fit into the first word.
 * This is only for testing the kernel.
 */
__host__ __device__ void insertEdge(uint src, uint dst, uint *graph)
{
    uint index = 1 << 12;
    while (graph[index] != UINT_MAX && graph[index+30] != 0)
        index += ELEMENT_WIDTH;
    if(graph[index] == UINT_MAX)
    for (size_t i = 0; i < ELEMENT_WIDTH - 1; i++)
    {
        graph[index + i] = 0;
    }

    graph[index] |= 1 << dst;
    graph[src] = index;
}

__global__ void kernel(int n, uint *A, uint *B, uint *C)
{
    // each warp gets a shared block for one access to global memory
    __shared__ uint _sh_[THREADS_PER_BLOCK / WARP_SIZE * 128];
    uint *const _shared_ = &_sh_[threadIdx.y * 128];
    for (uint src = blockIdx.x * blockDim.x + threadIdx.y; src < n; src += blockDim.x * gridDim.x)
    {
        uint index = A[src];
        do
        {
            uint bits = A[index + threadIdx.x];
            uint base = A[index + BASE];
            if (base == UINT_MAX)
                break;
            // create mask for threads w/ dst nodes, except last 2 (BASE & NEXT)
            uint nonEmptyThreads = __ballot_sync(0x3FFFFFFF, bits);
            const uint threadMask = 1 << threadIdx.x;
            const uint myMask = threadMask - 1;
            while (nonEmptyThreads)
            {
                // work through the nonEmptyThreads bits, get thread number of first thread w/ non empty bits
                int leastThread = __ffs(nonEmptyThreads) - 1;
                // remove lsb from nonEmptyThreads (iteration step)
                nonEmptyThreads &= (nonEmptyThreads - 1);
                // share current bits with all threads in warp
                uint current_bits = __shfl_sync(0x3FFFFFFF, bits, leastThread);
                
                // use the base and the word of the current thread's bits to calculate the target dst id
                uint var = base * 30 * 32 + 32 * leastThread + threadIdx.x;
                // check if this thread is looking at a dst node
                // uint bitActive = (var != 1U) && (current_bits & threadMask);
                uint bitActive = (current_bits & threadMask);
                // count threads that are looking at dst nodes
                uint threadsWithDstNode = __ballot_sync(0xFFFFFFFF, bitActive);
                uint numDstNodes = __popc(threadsWithDstNode);
                // calculate pos in shared mem, by counting prev threads that had a dst node
                uint pos = 0 + __popc(threadsWithDstNode & myMask);
                if (bitActive)
                {
                    _shared_[pos] = var;
                }
            }

            index = A[index + NEXT];
        } while (index != UINT_MAX);
    }
}

__host__ int run()
{
    // CUDA kernel to add elements of two arrays

    int N = 1 << 20;
    uint *pts, *prevPtsDiff, *currPtsDiff, *invCopy, *invStore, *invLoad;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&prevPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&currPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invCopy, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invStore, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invLoad, N * sizeof(uint1)));

    // set all values to UINT_MAX
    hipMemset(pts, UINT_MAX, N);
    hipMemset(prevPtsDiff, UINT_MAX, N);
    hipMemset(currPtsDiff, UINT_MAX, N);
    hipMemset(invCopy, UINT_MAX, N);
    hipMemset(invStore, UINT_MAX, N);
    hipMemset(invLoad, UINT_MAX, N);

    insertEdge(1, 0, pts);
    insertEdge(2, 1, invLoad);
    insertEdge(1, 3, invStore);
    insertEdge(3, 4, pts);

    // Launch kernel on 1M elements on the GPU

    dim3 numBlocks(1);
    dim3 threadsPerBlock(warpSize, 1024 / warpSize);
    kernel<<<numBlocks, threadsPerBlock>>>(5, invCopy, pts, pts);

    // Wait for GPU to finish before accessing on host
    checkCuda(hipDeviceSynchronize());

    // Free memory
    checkCuda(hipFree(pts));
    checkCuda(hipFree(prevPtsDiff));
    checkCuda(hipFree(currPtsDiff));
    checkCuda(hipFree(invCopy));
    checkCuda(hipFree(invStore));
    checkCuda(hipFree(invLoad));

    return 0;
}