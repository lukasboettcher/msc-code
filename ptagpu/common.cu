#include "hip/hip_runtime.h"
#include "common.cuh"

/**
 * getHeadIndex
 *
 * get the index of the first element for a given node
 *
 * \param src the node for which to get the head index
 *
 * \return index of the
 *
 */
__host__ __device__ size_t getHeadIndex(uint src, uint *graph)
{
    return 0;
}

__global__ void kernel(int n, uint *A, uint *B, uint *C)
{
}

__host__ int run()
{
    // CUDA kernel to add elements of two arrays

    int N = 1 << 8;
    uint *pts, *prevPtsDiff, *currPtsDiff, *invCopy, *invStore, *invLoad;

    // Allocate Unified Memory -- accessible from CPU or GPU
    checkCuda(hipMallocManaged(&pts, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&prevPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&currPtsDiff, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invCopy, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invStore, N * sizeof(uint1)));
    checkCuda(hipMallocManaged(&invLoad, N * sizeof(uint1)));

    getHeadIndex(0, pts);
    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        pts[i] = 1;
    }

    // Launch kernel on 1M elements on the GPU

    dim3 numBlocks(1);
    dim3 threadsPerBlock(warpSize, 1024 / warpSize);
    kernel<<<numBlocks, threadsPerBlock>>>(N, invCopy, pts, pts);

    // Wait for GPU to finish before accessing on host
    checkCuda(hipDeviceSynchronize());

    // Free memory
    checkCuda(hipFree(pts));
    checkCuda(hipFree(prevPtsDiff));
    checkCuda(hipFree(currPtsDiff));
    checkCuda(hipFree(invCopy));
    checkCuda(hipFree(invStore));
    checkCuda(hipFree(invLoad));

    return 0;
}