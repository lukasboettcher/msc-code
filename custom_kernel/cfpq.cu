#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <bits/stdc++.h>

using namespace std;

#define BLOCK_SIZE 32

#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char *const func, const char *const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
__global__ void gpu_matrix_mult(T *a, T *b, T *c, size_t m, size_t n, size_t k)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    T sum = 0;
    if (col < k && row < m)
    {
        for (size_t i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

template <typename T>
__device__ T get_lhs(T const a, T const b){
    T res = 0;

    // lsb   = a
    // lsb+1 = b
    // lsb+2 = c

    if (a & 1<<0 && b & 1<<1)
    {
        res |= 1<<2;
    }
    
    return res;
}

__device__ int d_changed;

template <typename T>
__global__ void closure_kernel(T *a, size_t V)
{
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    T sum = 0;
    if (col < V && row < V)
    {
        for (size_t i = 0; i < V; i++)
        {
            // sum += a[row * n + i] * b[i * k + col];
            sum |= get_lhs(a[row * V + i], a[i * V + col]);
            // sum |= a[row * V + i] & a[i * V + col];
        }
        a[row * V + col] = sum;
    }
}

template <typename T>
__global__ void closure_kernel_blocked(T *d_a, size_t n)
{
    __shared__ T tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ T tile_b[BLOCK_SIZE][BLOCK_SIZE];

    size_t row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    size_t col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    T tmp = 0;
    size_t idx;

    for (size_t sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if (idx >= n * n)
        {
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if (idx >= n * n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_a[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp |= get_lhs(tile_a[threadIdx.y][k], tile_b[k][threadIdx.x]);
            // tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if (row < n && col < n)
    {
        d_a[row * n + col] = tmp;
    }
}

template <typename T>
void printMatrix(T *m, size_t ld)
{
    for (size_t i = 0; i < ld; i++)
    {
        for (size_t j = 0; j < ld; j++)
        {
            T val = m[i * ld + j];
            // printbits(val);
            // putchar(' ');
            cout << val << "\t";
        }
        cout << endl;
    }
    cout << endl;
}

int main(int argc, char const *argv[])
{
    typedef unsigned char myType;
    srand(3333);

    size_t V{1<<16}, freeMem, totalMem;
    checkCuda(hipMemGetInfo(&freeMem, &totalMem));
    cout << "free memory:\t" << (double)freeMem/(1024*1024) << " MiB" << endl;
    cout << "needed memory:\t" << (double)(sizeof(myType) * V * V)/(1024*1024) << " MiB" << endl;
    

    // allocate memory in host RAM, h_cc is used to store CPU result
    // myType *h_a, *h_b, *h_c, *cpu_check;
    myType *h_a, *d_a;
    checkCuda(hipHostMalloc((void **)&h_a, sizeof(myType) * V * V));
    // checkCuda(hipHostMalloc((void **)&h_b, sizeof(myType) * V * V));
    // checkCuda(hipHostMalloc((void **)&h_c, sizeof(myType) * V * V));
    // checkCuda(hipHostMalloc((void **)&cpu_check, sizeof(myType) * V * V));

    // random initialize matrix A
    for (int i = 0; i < V; ++i)
    {
        for (int j = 0; j < V; ++j)
        {
            h_a[i * V + j] = rand();
            // h_b[i * V + j] = 0;
        }
    }

    // // random initialize matrix B
    // for (int i = 0; i < V; ++i)
    // {
    //     for (int j = 0; j < V; ++j)
    //     {
    //         h_b[i * V + j] = rand() %3;
    //     }
    // }

    // h_a[1] = 1;
    // h_a[3] = 2;

    // printMatrix(h_a, V);
    // printMatrix(h_b, V);

    hipEvent_t startEvent, stopEvent;
    float time{0.0f};

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    // // Allocate memory space on the device
    // myType *d_a, *d_b, *d_c;

    checkCuda(hipMalloc((void **)&d_a, sizeof(myType) * V * V));
    // checkCuda(hipMalloc((void **)&d_b, sizeof(myType) * V * V));
    // checkCuda(hipMalloc((void **)&d_c, sizeof(myType) * V * V));

    // copy matrix A and B from host to device memory
    checkCuda(hipMemcpy(d_a, h_a, sizeof(myType) * V * V, hipMemcpyHostToDevice));
    // checkCuda(hipMemcpy(d_b, h_b, sizeof(myType) * V * V, hipMemcpyHostToDevice));

    size_t grid_rows = (V + BLOCK_SIZE - 1) / BLOCK_SIZE;
    size_t grid_cols = (V + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    checkCuda(hipEventRecord(startEvent, 0));
    // gpu_square_matrix_mu<lt<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, V);
    // closure_kernel<<<dimGrid, dimBlock>>>(d_a, V);
    closure_kernel_blocked<<<dimGrid, dimBlock>>>(d_a, V);
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

    typeof(d_changed) h_changed;
    hipMemcpyFromSymbol(&h_changed, HIP_SYMBOL("d_changed"), sizeof(h_changed), 0, hipMemcpyDeviceToHost);

    // Transefr results from device to host
    checkCuda(hipMemcpy(h_a, d_a, sizeof(myType) * V * V, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());

    // printMatrix(h_a, V);

    cout << "time: " << time << endl;

    // free memory
    checkCuda(hipFree(d_a));
    // checkCuda(hipFree(d_b));
    // checkCuda(hipFree(d_c));
    checkCuda(hipHostFree(h_a));
    // checkCuda(hipHostFree(h_b));
    // checkCuda(hipHostFree(h_c));
    // checkCuda(hipHostFree(cpu_check));
    return 0;
}
